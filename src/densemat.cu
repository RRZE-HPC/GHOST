#include "hip/hip_runtime.h"
#include "ghost/config.h"
#undef GHOST_HAVE_MPI
#include "ghost/types.h"
#include "ghost/util.h"
#include "ghost/densemat.h"
#include "ghost/log.h"
#include "ghost/timing.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>
#include <sys/types.h>
#include <unistd.h>


#include "ghost/cu_complex.h"


#define THREADSPERBLOCK 256


template<typename T>  
__global__ static void cu_vaxpy_kernel(T *v1, T *v2, T *a, ghost_idx_t nrows, ghost_idx_t ncols, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_idx_t v;
        for (v=0; v<ncols; v++) {
            v1[v*nrowspadded+idx] = axpy<T,T>(v1[v*nrowspadded+idx],v2[v*nrowspadded+idx],a[v]);
        }
    }
}

template<typename T>  
__global__ static void cu_vaxpby_kernel(T *v1, T *v2, T *a, T *b, ghost_idx_t nrows, ghost_idx_t ncols, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_idx_t v;
        for (v=0; v<ncols; v++) {
            v1[v*nrowspadded+idx] = axpby<T>(v2[v*nrowspadded+idx],v1[v*nrowspadded+idx],a[v],b[v]);
        }
    }
}

template<typename T>  
__global__ static void cu_axpby_kernel(T *v1, T *v2, T a, T b, ghost_idx_t nrows, ghost_idx_t ncols, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_idx_t v;
        for (v=0; v<ncols; v++) {
            v1[v*nrowspadded+idx] = axpby<T>(v2[v*nrowspadded+idx],v1[v*nrowspadded+idx],a,b);
        }
    }
}

template<typename T>  
__global__ static void cu_vscale_kernel(T *vec, T *a, ghost_idx_t nrows, ghost_idx_t ncols, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_idx_t v;
        for (v=0; v<ncols; v++) {
            vec[v*nrowspadded+idx] = scale<T>(a[v],vec[v*nrowspadded+idx]);
        }
    }
}

template<typename T>  
__global__ static void cu_fromscalar_kernel(T *vec, T a, ghost_idx_t nrows, ghost_idx_t ncols, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_idx_t v;
        for (v=0; v<ncols; v++) {
            vec[v*nrowspadded+idx] = a;
        }
    }
}

extern "C" ghost_error_t ghost_vec_cu_vaxpy(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a)
{
    void *d_a;
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,v1->traits.datatype);
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_a,v1->traits.ncols*sizeofdt));
    ghost_cu_upload(d_a,a,v1->traits.ncols*sizeofdt);
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot VAXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }

    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpy_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,(hipDoubleComplex *)d_a,v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpy_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,(hipFloatComplex *)d_a,v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpy_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((double *)v1->cu_val, (double *)v2->cu_val,(double *)d_a,v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpy_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((float *)v1->cu_val, (float *)v2->cu_val,(float *)d_a,v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        }
    }
    return GHOST_SUCCESS;
}
    
extern "C" ghost_error_t ghost_vec_cu_vaxpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    void *d_a;
    void *d_b;
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,v1->traits.datatype);
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_a,v1->traits.ncols*sizeofdt)); //TODO goto and free
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_b,v1->traits.ncols*sizeofdt));
    ghost_cu_upload(d_b,b,v1->traits.ncols*sizeofdt);
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot VAXPBY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,(hipDoubleComplex *)d_a,(hipDoubleComplex *)d_b,
                 v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,(hipFloatComplex *)d_a,(hipFloatComplex *)d_b,
                 v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)v1->cu_val, (double *)v2->cu_val,(double *)d_a,(double *)d_b,
                    v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpby_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (float *)v1->cu_val, (float *)v2->cu_val,(float *)d_a,(float *)d_b,
                 v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_vec_cu_dotprod(ghost_densemat_t *vec, ghost_densemat_t *vec2, void *res)
{
    if (vec->traits.datatype != vec2->traits.datatype)
    {
        ERROR_LOG("Cannot DOT vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,vec->traits.datatype);
   
    hipblasHandle_t ghost_cublas_handle;
    GHOST_CALL_RETURN(ghost_cu_cublas_handle(&ghost_cublas_handle)); 
    ghost_idx_t v;
    for (v=0; v<vec->traits.ncols; v++)
    {
        char *v1 = &((char *)(vec->cu_val))[v*vec->traits.nrowspadded*sizeofdt];
        char *v2 = &((char *)(vec2->cu_val))[v*vec->traits.nrowspadded*sizeofdt];
        if (vec->traits.datatype & GHOST_DT_COMPLEX)
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_RETURN(hipblasZdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipDoubleComplex *)v1,1,(const hipDoubleComplex *)v2,1,&((hipDoubleComplex *)res)[v]));
            } 
            else 
            {
                CUBLAS_CALL_RETURN(hipblasCdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipFloatComplex *)v1,1,(const hipFloatComplex *)v2,1,&((hipFloatComplex *)res)[v]));
            }
        }
        else
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_RETURN(hipblasDdot(ghost_cublas_handle,vec->traits.nrows,
                            (const double *)v1,1,(const double *)v2,1,&((double *)res)[v]));
            } 
            else 
            {
                CUBLAS_CALL_RETURN(hipblasSdot(ghost_cublas_handle,vec->traits.nrows,
                            (const float *)v1,1,(const float *)v2,1,&((float *)res)[v]));
            }
        }
    }
    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_vec_cu_axpy(ghost_densemat_t *vec, ghost_densemat_t *vec2, void *a)
{
    if (vec->traits.datatype != vec2->traits.datatype)
    {
        ERROR_LOG("Cannot AXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    hipblasHandle_t ghost_cublas_handle;
    GHOST_CALL_RETURN(ghost_cu_cublas_handle(&ghost_cublas_handle)); 
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CUBLAS_CALL_RETURN(hipblasZaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const hipDoubleComplex *)a,
                        (const hipDoubleComplex *)vec2->cu_val,1,
                        (hipDoubleComplex *)vec->cu_val,1));
        } 
        else 
        {
            CUBLAS_CALL_RETURN(hipblasCaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const hipFloatComplex *)a,
                        (const hipFloatComplex *)vec2->cu_val,1,
                        (hipFloatComplex *)vec->cu_val,1));
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CUBLAS_CALL_RETURN(hipblasDaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const double *)a,
                        (const double *)vec2->cu_val,1,
                        (double *)vec->cu_val,1));
        } 
        else 
        {
            CUBLAS_CALL_RETURN(hipblasSaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const float *)a,
                        (const float *)vec2->cu_val,1,
                        (float *)vec->cu_val,1));
        }
    }
    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_vec_cu_axpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot AXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,*((hipDoubleComplex *)a),*((hipDoubleComplex *)b),
                 v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_axpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,*((hipFloatComplex *)a),*((hipFloatComplex *)b),
                 v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((double *)v1->cu_val, (double *)v2->cu_val,*((double *)a),*((double *)b),
                 v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_axpby_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((float *)v1->cu_val, (float *)v2->cu_val,*((float *)a),*((float *)b),
                 v1->traits.nrows,v1->traits.ncols,v1->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_vec_cu_scale(ghost_densemat_t *vec, void *a)
{
    hipblasHandle_t ghost_cublas_handle;
    GHOST_CALL_RETURN(ghost_cu_cublas_handle(&ghost_cublas_handle)); 
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CUBLAS_CALL_RETURN(hipblasZscal(ghost_cublas_handle,vec->traits.nrows,
                        (const hipDoubleComplex *)a,
                        (hipDoubleComplex *)vec->cu_val,1));
        } 
        else 
        {
            CUBLAS_CALL_RETURN(hipblasCscal(ghost_cublas_handle,vec->traits.nrows,
                        (const hipFloatComplex *)a,
                        (hipFloatComplex *)vec->cu_val,1));
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CUBLAS_CALL_RETURN(hipblasDscal(ghost_cublas_handle,vec->traits.nrows,
                        (const double *)a,
                        (double *)vec->cu_val,1));
        } 
        else 
        {
            CUBLAS_CALL_RETURN(hipblasSscal(ghost_cublas_handle,vec->traits.nrows,
                        (const float *)a,
                        (float *)vec->cu_val,1));
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_vec_cu_vscale(ghost_densemat_t *vec, void *a)
{
    void *d_a;
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,vec->traits.datatype);
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_a,vec->traits.ncols*sizeofdt));
    ghost_cu_upload(d_a,a,vec->traits.ncols*sizeofdt);
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vec->cu_val, (hipDoubleComplex *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_vscale_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vec->cu_val, (hipFloatComplex *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vec->cu_val, (double *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_vscale_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vec->cu_val, (float *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_vec_cu_fromScalar(ghost_densemat_t *vec, void *a)
{
    ghost_vec_malloc(vec);
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vec->cu_val, *(hipDoubleComplex *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_fromscalar_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vec->cu_val, *(hipFloatComplex *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vec->cu_val, *(double *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_fromscalar_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vec->cu_val, *(float *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_vec_cu_fromRand(ghost_densemat_t *vec)
{
    long pid = getpid();
    double time;
    ghost_timing_wcmilli(&time);
    ghost_vec_malloc(vec);
    hiprandGenerator_t gen;
    CURAND_CALL_RETURN(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL_RETURN(hiprandSetPseudoRandomGeneratorSeed(gen,ghost_hash(int(time),clock(),pid)));

    ghost_idx_t v;
    for (v=0; v<vec->traits.ncols; v++)
    {
        if (vec->traits.datatype & GHOST_DT_COMPLEX)
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CURAND_CALL_RETURN(hiprandGenerateUniformDouble(gen,
                            &((double *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows*2));
            } 
            else 
            {
                CURAND_CALL_RETURN(hiprandGenerateUniform(gen,
                            &((float *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows*2));
            }
        }
        else
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CURAND_CALL_RETURN(hiprandGenerateUniformDouble(gen,
                            &((double *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows));
            } 
            else 
            {
                CURAND_CALL_RETURN(hiprandGenerateUniform(gen,
                            &((float *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows));
            }
        }
    }

    return GHOST_SUCCESS;
}
