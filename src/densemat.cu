#include "hip/hip_runtime.h"
#include "ghost/config.h"
#undef GHOST_HAVE_MPI
#include "ghost/types.h"
#include "ghost/util.h"
#include "ghost/densemat_cm.h"
#include "ghost/log.h"
#include "ghost/timing.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>
#include <sys/types.h>
#include <unistd.h>

#include "ghost/cu_complex.h"


#define THREADSPERBLOCK 256


template<typename T>  
__global__ static void cu_vaxpy_kernel(T *v1, T *v2, T *a, ghost_idx_t nrows, char *rowmask, ghost_idx_t ncols, char *colmask, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        if (rowmask[idx]) {
            ghost_idx_t v;
            for (v=0; v<ncols; v++) {
                if (colmask[v]) {
                    v1[v*nrowspadded+idx] = axpy<T,T>(v1[v*nrowspadded+idx],v2[v*nrowspadded+idx],a[v]);
                }
            }
        }
    }
}

template<typename T>  
__global__ static void cu_vaxpby_kernel(T *v1, T *v2, T *a, T *b, ghost_idx_t nrows, char *rowmask, ghost_idx_t ncols, char *colmask, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        if (rowmask[idx]) {
            ghost_idx_t v;
            for (v=0; v<ncols; v++) {
                if (colmask[v]) {
                    v1[v*nrowspadded+idx] = axpby<T>(v2[v*nrowspadded+idx],v1[v*nrowspadded+idx],a[v],b[v]);
                }
            }
        }
    }
}

template<typename T>  
__global__ static void cu_axpby_kernel(T *v1, T *v2, T a, T b, ghost_idx_t nrows, char *rowmask, ghost_idx_t ncols, char *colmask, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        if (rowmask[idx]) {
            ghost_idx_t v;
            for (v=0; v<ncols; v++) {
                if (colmask[v]) {
                    v1[v*nrowspadded+idx] = axpby<T>(v2[v*nrowspadded+idx],v1[v*nrowspadded+idx],a,b);
                }
            }
        }
    }
}

template<typename T>  
__global__ static void cu_scale_kernel(T *vec, T a, ghost_idx_t nrows, char *rowmask, ghost_idx_t ncols, char *colmask, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        if (rowmask[idx]) {
            ghost_idx_t v;
            for (v=0; v<ncols; v++) {
                if (colmask[v]) {
                    vec[v*nrowspadded+idx] = scale<T>(a,vec[v*nrowspadded+idx]);
                }
            }
        }
    }
}

template<typename T>  
__global__ static void cu_vscale_kernel(T *vec, T *a, ghost_idx_t nrows, char *rowmask, ghost_idx_t ncols, char *colmask, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int c;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        if (rowmask[idx]) {
            ghost_idx_t v;
            for (c=0,v=0; v<ncols; v++) {
                if (colmask[v]) {
                    vec[v*nrowspadded+idx] = scale<T>(a[c],vec[v*nrowspadded+idx]);
                    c++;
                }
            }
        }
    }
}

template<typename T>  
__global__ static void cu_fromscalar_kernel(T *vec, T a, ghost_idx_t nrows, char *rowmask, ghost_idx_t ncols, char *colmask, ghost_idx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        if (rowmask[idx]) {
            ghost_idx_t v;
            for (v=0; v<ncols; v++) {
                if (colmask[v]) {
                    vec[v*nrowspadded+idx] = a;
                }
            }
        }
    }
}

extern "C" ghost_error_t ghost_densemat_cm_cu_vaxpy(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a)
{
    void *d_a;
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,v1->traits.datatype);
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_a,v1->traits.ncols*sizeofdt));
    ghost_cu_upload(d_a,a,v1->traits.ncols*sizeofdt);
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot VAXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    char colfield[v1->traits.ncolsorig];
    char rowfield[v1->traits.nrowsorig];

    char *cucolfield, *curowfield;
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,v1->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,v1->traits.nrowsorig));

    ghost_densemat_mask2charfield(v1->cumask,v1->traits.ncolsorig,colfield);
    ghost_densemat_mask2charfield(v1->mask,v1->traits.nrowsorig,rowfield);

    GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,v1->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,v1->traits.nrowsorig));

    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpy_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,(hipDoubleComplex *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpy_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,(hipFloatComplex *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpy_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((double *)v1->cu_val, (double *)v2->cu_val,(double *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpy_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((float *)v1->cu_val, (float *)v2->cu_val,(float *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        }
    }
    return GHOST_SUCCESS;
}
    
extern "C" ghost_error_t ghost_densemat_cm_cu_vaxpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    void *d_a;
    void *d_b;
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,v1->traits.datatype);
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_a,v1->traits.ncols*sizeofdt)); //TODO goto and free
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_b,v1->traits.ncols*sizeofdt));
    ghost_cu_upload(d_b,b,v1->traits.ncols*sizeofdt);
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot VAXPBY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    char colfield[v1->traits.ncolsorig];
    char rowfield[v1->traits.nrowsorig];

    char *cucolfield, *curowfield;
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,v1->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,v1->traits.nrowsorig));

    ghost_densemat_mask2charfield(v1->cumask,v1->traits.ncolsorig,colfield);
    ghost_densemat_mask2charfield(v1->mask,v1->traits.nrowsorig,rowfield);

    GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,v1->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,v1->traits.nrowsorig));
    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,(hipDoubleComplex *)d_a,(hipDoubleComplex *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,(hipFloatComplex *)d_a,(hipFloatComplex *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                 (double *)v1->cu_val, (double *)v2->cu_val,(double *)d_a,(double *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_vaxpby_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (float *)v1->cu_val, (float *)v2->cu_val,(float *)d_a,(float *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_densemat_cm_cu_dotprod(ghost_densemat_t *vec, void *res, ghost_densemat_t *vec2)
{
    if (vec->traits.datatype != vec2->traits.datatype)
    {
        ERROR_LOG("Cannot DOT vectors with different data types (%s and %s)",ghost_datatype_string(vec->traits.datatype),ghost_datatype_string(vec2->traits.datatype));
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,vec->traits.datatype);
    ghost_densemat_t *vecclone;
    ghost_densemat_t *vec2clone;

    WARNING_LOG("Cloning (and compressing) vectors before dotproduct");
    vec->clone(vec,&vecclone,vec->traits.nrows,0,vec->traits.ncols,0);
    vec2->clone(vec2,&vec2clone,vec2->traits.nrows,0,vec2->traits.ncols,0);
  
     
    hipblasHandle_t ghost_cublas_handle;
    GHOST_CALL_RETURN(ghost_cu_cublas_handle(&ghost_cublas_handle)); 
    ghost_idx_t v;
    for (v=0; v<vecclone->traits.ncols; v++)
    {
        char *v1 = &((char *)(vecclone->cu_val))[v*vecclone->traits.nrowspadded*sizeofdt];
        char *v2 = &((char *)(vec2clone->cu_val))[v*vec2clone->traits.nrowspadded*sizeofdt];
        if (vec->traits.datatype & GHOST_DT_COMPLEX)
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_RETURN(hipblasZdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipDoubleComplex *)v1,1,(const hipDoubleComplex *)v2,1,&((hipDoubleComplex *)res)[v]));
            } 
            else 
            {
                CUBLAS_CALL_RETURN(hipblasCdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipFloatComplex *)v1,1,(const hipFloatComplex *)v2,1,&((hipFloatComplex *)res)[v]));
            }
        }
        else
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_RETURN(hipblasDdot(ghost_cublas_handle,vec->traits.nrows,
                            (const double *)v1,1,(const double *)v2,1,&((double *)res)[v]));
            } 
            else 
            {
                CUBLAS_CALL_RETURN(hipblasSdot(ghost_cublas_handle,vec->traits.nrows,
                            (const float *)v1,1,(const float *)v2,1,&((float *)res)[v]));
            }
        }
    }
    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_densemat_cm_cu_axpy(ghost_densemat_t *vec, ghost_densemat_t *vec2, void *a)
{
    if (vec->traits.datatype != vec2->traits.datatype)
    {
        ERROR_LOG("Cannot AXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];

    char *cucolfield, *curowfield;
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig));

    ghost_densemat_mask2charfield(vec->cumask,vec->traits.ncolsorig,colfield);
    ghost_densemat_mask2charfield(vec->mask,vec->traits.nrowsorig,rowfield);

    GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig));

    
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            const hipDoubleComplex one = make_hipDoubleComplex(1.,1.);
            cu_axpby_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipDoubleComplex *)vec->cu_val, (hipDoubleComplex *)vec2->cu_val,*((hipDoubleComplex *)a),one,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            const hipFloatComplex one = make_hipFloatComplex(1.,1.);
            cu_axpby_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipFloatComplex *)vec->cu_val, (hipFloatComplex *)vec2->cu_val,*((hipFloatComplex *)a),one,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((double *)vec->cu_val, (double *)vec2->cu_val,*((double *)a),(double)1.,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_axpby_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((float *)vec->cu_val, (float *)vec2->cu_val,*((float *)a),(float)1.,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }

    
    /*hipblasHandle_t ghost_cublas_handle;
    GHOST_CALL_RETURN(ghost_cu_cublas_handle(&ghost_cublas_handle)); 
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CUBLAS_CALL_RETURN(hipblasZaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const hipDoubleComplex *)a,
                        (const hipDoubleComplex *)vec2->cu_val,1,
                        (hipDoubleComplex *)vec->cu_val,1));
        } 
        else 
        {
            CUBLAS_CALL_RETURN(hipblasCaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const hipFloatComplex *)a,
                        (const hipFloatComplex *)vec2->cu_val,1,
                        (hipFloatComplex *)vec->cu_val,1));
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CUBLAS_CALL_RETURN(hipblasDaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const double *)a,
                        (const double *)vec2->cu_val,1,
                        (double *)vec->cu_val,1));
        } 
        else 
        {
            CUBLAS_CALL_RETURN(hipblasSaxpy(ghost_cublas_handle,vec->traits.nrows,
                        (const float *)a,
                        (const float *)vec2->cu_val,1,
                        (float *)vec->cu_val,1));
        }
    }*/
    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_densemat_cm_cu_axpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot AXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    char colfield[v1->traits.ncolsorig];
    char rowfield[v1->traits.nrowsorig];

    char *cucolfield, *curowfield;
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,v1->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,v1->traits.nrowsorig));

    ghost_densemat_mask2charfield(v1->cumask,v1->traits.ncolsorig,colfield);
    ghost_densemat_mask2charfield(v1->mask,v1->traits.nrowsorig,rowfield);

    GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,v1->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,v1->traits.nrowsorig));
    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,*((hipDoubleComplex *)a),*((hipDoubleComplex *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_axpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,*((hipFloatComplex *)a),*((hipFloatComplex *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((double *)v1->cu_val, (double *)v2->cu_val,*((double *)a),*((double *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        } 
        else 
        {
            cu_axpby_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((float *)v1->cu_val, (float *)v2->cu_val,*((float *)a),*((float *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_densemat_cm_cu_scale(ghost_densemat_t *vec, void *a)
{
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];

    char *cucolfield, *curowfield;
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig));

    ghost_densemat_mask2charfield(vec->cumask,vec->traits.ncolsorig,colfield);
    ghost_densemat_mask2charfield(vec->mask,vec->traits.nrowsorig,rowfield);

    GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig));

    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_scale_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vec->cu_val, *(hipDoubleComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_scale_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vec->cu_val, *(hipFloatComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_scale_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vec->cu_val, *(double *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_scale_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vec->cu_val, *(float *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_densemat_cm_cu_vscale(ghost_densemat_t *vec, void *a)
{
    void *d_a;
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,vec->traits.datatype);
    GHOST_CALL_RETURN(ghost_cu_malloc(&d_a,vec->traits.ncols*sizeofdt));
    ghost_cu_upload(d_a,a,vec->traits.ncols*sizeofdt);
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];

    char *cucolfield, *curowfield;
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig));

    ghost_densemat_mask2charfield(vec->cumask,vec->traits.ncolsorig,colfield);
    ghost_densemat_mask2charfield(vec->mask,vec->traits.nrowsorig,rowfield);

    GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig));

    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vec->cu_val, (hipDoubleComplex *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_vscale_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vec->cu_val, (hipFloatComplex *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vec->cu_val, (double *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_vscale_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vec->cu_val, (float *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_densemat_cm_cu_fromScalar(ghost_densemat_t *vec, void *a)
{
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];

    char *cucolfield, *curowfield;
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig));

    ghost_densemat_mask2charfield(vec->cumask,vec->traits.ncolsorig,colfield);
    ghost_densemat_mask2charfield(vec->mask,vec->traits.nrowsorig,rowfield);

    GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig));
    GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig));
    ghost_densemat_cm_malloc(vec);
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vec->cu_val, *(hipDoubleComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_fromscalar_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vec->cu_val, *(hipFloatComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vec->cu_val, *(double *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        } 
        else 
        {
            cu_fromscalar_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vec->cu_val, *(float *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->traits.nrowspadded);
        }
    }

    return GHOST_SUCCESS;
}

extern "C" ghost_error_t ghost_densemat_cm_cu_fromRand(ghost_densemat_t *vec)
{
    long pid = getpid();
    double time;
    ghost_timing_wcmilli(&time);
    ghost_densemat_cm_malloc(vec);
    hiprandGenerator_t gen;
    CURAND_CALL_RETURN(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL_RETURN(hiprandSetPseudoRandomGeneratorSeed(gen,ghost_hash(int(time),clock(),pid)));

    ghost_idx_t v;
    for (v=0; v<vec->traits.ncols; v++)
    {
        if (vec->traits.datatype & GHOST_DT_COMPLEX)
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CURAND_CALL_RETURN(hiprandGenerateUniformDouble(gen,
                            &((double *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows*2));
            } 
            else 
            {
                CURAND_CALL_RETURN(hiprandGenerateUniform(gen,
                            &((float *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows*2));
            }
        }
        else
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CURAND_CALL_RETURN(hiprandGenerateUniformDouble(gen,
                            &((double *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows));
            } 
            else 
            {
                CURAND_CALL_RETURN(hiprandGenerateUniform(gen,
                            &((float *)(vec->cu_val))[v*vec->traits.nrowspadded],
                            vec->traits.nrows));
            }
        }
    }

    return GHOST_SUCCESS;
}
