#include "hip/hip_runtime.h"
#include "ghost/config.h"
#include "ghost/types.h"
#include "ghost/log.h"
#include "ghost/cu_util.h"
#include "ghost/cu_complex.h"
#include "ghost/util.h"
#include "ghost/sparsemat.h"

#include <complex.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

typedef hipsparseStatus_t (*cusparse_sell1_spmv_kernel_t) (hipsparseHandle_t handle, hipsparseOperation_t transA, 
        int m, int n, int nnz, const void           *alpha, 
        const hipsparseMatDescr_t descrA, 
        const void           *csrValA, 
        const void *csrRowPtrA, const void *csrColIndA,
        const void           *x, const void           *beta, 
        void           *y);

typedef hipsparseStatus_t (*cusparse_sell1_spmmv_cm_kernel_t) (hipsparseHandle_t handle, hipsparseOperation_t transA, 
        int m, int n, int k, int nnz, const void           *alpha, 
        const hipsparseMatDescr_t descrA, 
        const void           *csrValA, 
        const void *csrRowPtrA, const void *csrColIndA,
        const void           *x, int ldx, const void           *beta, 
        void           *y, int ldy);

typedef hipsparseStatus_t (*cusparse_sell1_spmmv_rm_kernel_t) (hipsparseHandle_t handle, hipsparseOperation_t transA,
        hipsparseOperation_t transB,
        int m, int n, int k, int nnz, const void           *alpha, 
        const hipsparseMatDescr_t descrA, 
        const void           *csrValA, 
        const void *csrRowPtrA, const void *csrColIndA,
        const void           *x, int ldx, const void           *beta, 
        void           *y, int ldy);

    template<typename dt1, typename dt2>
static ghost_error ghost_cu_sell1spmv_tmpl(ghost_sparsemat *mat, ghost_densemat * lhs, ghost_densemat * rhs, ghost_spmv_opts traits, cusparse_sell1_spmv_kernel_t sell1kernel)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    hipsparseHandle_t cusparse_handle;
    hipsparseMatDescr_t descr;

    hipsparseCreateMatDescr(&descr);
    GHOST_CALL_RETURN(ghost_cu_cusparse_handle(&cusparse_handle));

    dt2 * __attribute__((unused)) localdot = NULL;
    dt1 * __attribute__((unused)) shift = NULL, scale, beta, __attribute__((unused)) sdelta, __attribute__((unused)) seta;
    ghost_densemat * __attribute__((unused)) z = NULL;

    one<dt1>(scale);

    GHOST_SPMV_PARSE_TRAITS(traits,scale,beta,shift,localdot,z,sdelta,seta,dt2,dt1);

    if (traits.flags & GHOST_SPMV_AXPY) {
        one<dt1>(beta);
    } else if (!(traits.flags & GHOST_SPMV_AXPBY)) {
        zero<dt1>(beta);
    }
     
    CUSPARSE_CALL_RETURN(sell1kernel(cusparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,mat->nrows,rhs->traits.nrowshalo,mat->nnz,&scale,descr,(dt1 *)mat->cu_val, mat->cu_chunkStart, mat->cu_col, (dt1 *)rhs->cu_val, &beta, (dt1 *)lhs->cu_val));
   
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);
    return GHOST_SUCCESS;
}
    
    template<typename dt1, typename dt2>
static ghost_error ghost_cu_sell1spmmv_cm_tmpl(ghost_sparsemat *mat, ghost_densemat * lhs, ghost_densemat * rhs, ghost_spmv_opts traits, cusparse_sell1_spmmv_cm_kernel_t sell1kernel)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    hipsparseHandle_t cusparse_handle;
    hipsparseMatDescr_t descr;

    hipsparseCreateMatDescr(&descr);
    GHOST_CALL_RETURN(ghost_cu_cusparse_handle(&cusparse_handle));

    dt2 * __attribute__((unused)) localdot = NULL;
    dt1 * __attribute__((unused)) shift = NULL, scale, beta, __attribute__((unused)) sdelta, __attribute__((unused)) seta;
    ghost_densemat * __attribute__((unused)) z = NULL;

    one<dt1>(scale);

    GHOST_SPMV_PARSE_TRAITS(traits,scale,beta,shift,localdot,z,sdelta,seta,dt2,dt1);
    
    if (traits.flags & GHOST_SPMV_AXPY) {
        one<dt1>(beta);
    } else if (!(traits.flags & GHOST_SPMV_AXPBY)) {
        zero<dt1>(beta);
    }
    CUSPARSE_CALL_RETURN(sell1kernel(cusparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,mat->nrows,rhs->traits.ncols,rhs->traits.nrowshalo,mat->nnz,&scale,descr,(dt1 *)mat->cu_val, mat->cu_chunkStart, mat->cu_col, (dt1 *)rhs->cu_val, rhs->stride, &beta, (dt1 *)lhs->cu_val, lhs->stride));

    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);
    return GHOST_SUCCESS;
}

    template<typename dt1, typename dt2>
static ghost_error ghost_cu_sell1spmmv_rm_tmpl(ghost_sparsemat *mat, ghost_densemat * lhs, ghost_densemat * rhs, ghost_spmv_opts traits, cusparse_sell1_spmmv_rm_kernel_t sell1kernel)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    hipsparseHandle_t cusparse_handle;
    hipsparseMatDescr_t descr;

    hipsparseCreateMatDescr(&descr);
    GHOST_CALL_RETURN(ghost_cu_cusparse_handle(&cusparse_handle));

    dt2 * __attribute__((unused)) localdot = NULL;
    dt1 * __attribute__((unused)) shift = NULL, scale, beta, __attribute__((unused)) sdelta, __attribute__((unused)) seta;
    ghost_densemat * __attribute__((unused)) z = NULL;

    one<dt1>(scale);

    GHOST_SPMV_PARSE_TRAITS(traits,scale,beta,shift,localdot,z,sdelta,seta,dt2,dt1);

    if (traits.flags & GHOST_SPMV_AXPY) {
        one<dt1>(beta);
    } else if (!(traits.flags & GHOST_SPMV_AXPBY)) {
        zero<dt1>(beta);
    }
    
    CUSPARSE_CALL_RETURN(sell1kernel(cusparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_TRANSPOSE,mat->nrows,rhs->traits.ncols,rhs->traits.nrowshalo,mat->nnz,&scale,descr,(dt1 *)mat->cu_val, mat->cu_chunkStart, mat->cu_col, (dt1 *)rhs->cu_val, rhs->stride, &beta, (dt1 *)lhs->cu_val,lhs->stride));
    
    
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);
    return GHOST_SUCCESS;
}
    
    template<typename dt1, typename dt2>
static ghost_error ghost_cu_sell1spmv_augfunc_tmpl(ghost_densemat * lhs, ghost_densemat * rhs, ghost_spmv_opts traits)
{
    dt2 *localdot = NULL;
    dt1 *shift = NULL, scale, __attribute__((unused)) beta, sdelta, seta;
    ghost_densemat *z = NULL;

    one<dt1>(scale); //required because we need scale for SHIFT (i.e., even if we don't SCALE)
    GHOST_SPMV_PARSE_TRAITS(traits,scale,beta,shift,localdot,z,sdelta,seta,dt2,dt1);

    if (traits.flags & (GHOST_SPMV_SHIFT|GHOST_SPMV_VSHIFT)) {
        PERFWARNING_LOG("Shift will not be applied on-the-fly!");
        dt2 minusshift[rhs->traits.ncols];
        ghost_lidx col;
        if (traits.flags & GHOST_SPMV_SHIFT) {
            for (col=0; col<rhs->traits.ncols; col++) {
                minusshift[col] = -1.*(*(dt2 *)&scale)*(*(dt2 *)shift);
            }
        } else {
            for (col=0; col<rhs->traits.ncols; col++) {
                minusshift[col] = -1.*(*(dt2 *)&scale)*(((dt2 *)shift)[col]);
            }
        }
        lhs->vaxpy(lhs,rhs,minusshift);
    }
    
    if (traits.flags & GHOST_SPMV_DOT) {
        PERFWARNING_LOG("Dot product computation will be not be done on-the-fly!");
        memset(localdot,0,lhs->traits.ncols*3*sizeof(dt1));
        if (traits.flags & GHOST_SPMV_DOT_YY) {
            lhs->localdot_vanilla(lhs,&localdot[0],lhs);
        }
        if (traits.flags & GHOST_SPMV_DOT_XY) {
            rhs->localdot_vanilla(rhs,&localdot[lhs->traits.ncols],lhs);
        }
        if (traits.flags & GHOST_SPMV_DOT_XX) {
            rhs->localdot_vanilla(rhs,&localdot[2*lhs->traits.ncols],rhs);
        }
            
    }
    if (traits.flags & GHOST_SPMV_CHAIN_AXPBY) {
        PERFWARNING_LOG("AXPBY will not be done on-the-fly!");
        z->axpby(z,lhs,&seta,&sdelta);
    }
   
    return GHOST_SUCCESS; 
}

ghost_error ghost_cu_sell1_spmv_selector(ghost_densemat * lhs_in, ghost_sparsemat *mat, ghost_densemat * rhs_in, ghost_spmv_opts traits)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    
    ghost_error ret = GHOST_SUCCESS;
    ghost_densemat *lhs, *rhs;
    
    if (mat->traits.datatype != lhs_in->traits.datatype) {
        ERROR_LOG("Mixed data types not implemented!");
        ret = GHOST_ERR_NOT_IMPLEMENTED;
        goto err;
    }
    if ((lhs_in->traits.flags & GHOST_DENSEMAT_SCATTERED) || (lhs_in->traits.storage == GHOST_DENSEMAT_ROWMAJOR)) {
        PERFWARNING_LOG("Cloning lhs");
        if (lhs_in->traits.flags & GHOST_DENSEMAT_SCATTERED) {
            PERFWARNING_LOG("Cloning and compressing lhs before operation because it is scattered");
        }
        if (lhs_in->traits.storage == GHOST_DENSEMAT_ROWMAJOR) {
            PERFWARNING_LOG("Cloning and transposing lhs before operation because it is row-major");
        }
        ghost_densemat_traits lhstraits = lhs_in->traits;
        lhstraits.location = GHOST_LOCATION_DEVICE;
        lhstraits.storage = GHOST_DENSEMAT_COLMAJOR;
        lhstraits.flags &= (ghost_densemat_flags)(~GHOST_DENSEMAT_VIEW);
        GHOST_CALL_GOTO(ghost_densemat_create(&lhs,NULL,lhstraits),err,ret);
        GHOST_CALL_GOTO(lhs->fromVec(lhs,lhs_in,0,0),err,ret);
    } else {
        lhs = lhs_in;
    }
    if ((rhs_in->traits.flags & GHOST_DENSEMAT_SCATTERED) || ((rhs_in->traits.ncols == 1) && (rhs_in->traits.storage == GHOST_DENSEMAT_ROWMAJOR))) {
        PERFWARNING_LOG("Cloning rhs");
        if (rhs_in->traits.flags & GHOST_DENSEMAT_SCATTERED) {
            PERFWARNING_LOG("Cloning and compressing rhs before operation because it is scattered");
        }
        if ((rhs_in->traits.ncols == 1) && (rhs_in->traits.storage == GHOST_DENSEMAT_ROWMAJOR)) {
            PERFWARNING_LOG("Cloning and transposing rhs before operation because it is row-major");
        }
        ghost_densemat_traits rhstraits = rhs_in->traits;
        rhstraits.location = GHOST_LOCATION_DEVICE;
        rhstraits.storage = GHOST_DENSEMAT_COLMAJOR;
        rhstraits.flags &= (ghost_densemat_flags)(~GHOST_DENSEMAT_VIEW);
        GHOST_CALL_GOTO(ghost_densemat_create(&rhs,NULL,rhstraits),err,ret);
        GHOST_CALL_GOTO(rhs->fromVec(rhs,rhs_in,0,0),err,ret);
        ghost_lidx nhalo = rhs->traits.nrowshalo - rhs->traits.nrowspadded;
        GHOST_CALL_GOTO(ghost_cu_memtranspose(nhalo,rhs->traits.ncols,&rhs->cu_val[rhs->traits.nrowspadded*rhs->elSize],rhs->stride,&rhs_in->cu_val[rhs_in->traits.nrowspadded*rhs_in->stride*rhs_in->elSize],rhs_in->stride,rhs->traits.datatype),err,ret);
    } else {
        rhs = rhs_in;
    }


    if (lhs->traits.ncols == 1) {
        if (mat->traits.datatype & GHOST_DT_DOUBLE) {
            if (mat->traits.datatype & GHOST_DT_REAL) {
                GHOST_CALL_GOTO((ghost_cu_sell1spmv_tmpl<double,double>(mat,lhs,rhs,traits,(cusparse_sell1_spmv_kernel_t)hipsparseDcsrmv)),err,ret);
            } else {
                GHOST_CALL_GOTO((ghost_cu_sell1spmv_tmpl<hipDoubleComplex,complex double>(mat,lhs,rhs,traits,(cusparse_sell1_spmv_kernel_t)hipsparseZcsrmv)),err,ret);
            }
        } else {
            if (mat->traits.datatype & GHOST_DT_REAL) {
                GHOST_CALL_GOTO((ghost_cu_sell1spmv_tmpl<float,float>(mat,lhs,rhs,traits,(cusparse_sell1_spmv_kernel_t)hipsparseScsrmv)),err,ret);
            } else {
                GHOST_CALL_GOTO((ghost_cu_sell1spmv_tmpl<hipFloatComplex,complex float>(mat,lhs,rhs,traits,(cusparse_sell1_spmv_kernel_t)hipsparseCcsrmv)),err,ret);
            }
        }
    } else if (rhs->traits.storage == GHOST_DENSEMAT_COLMAJOR) {
        INFO_LOG("Calling col-major cuSparse CRS SpMMV");
        if (mat->traits.datatype & GHOST_DT_DOUBLE) {
            if (mat->traits.datatype & GHOST_DT_REAL) {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_cm_tmpl<double,double>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_cm_kernel_t)hipsparseDcsrmm)),err,ret);
            } else {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_cm_tmpl<hipDoubleComplex,complex double>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_cm_kernel_t)hipsparseZcsrmm)),err,ret);
            }
        } else {
            if (mat->traits.datatype & GHOST_DT_REAL) {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_cm_tmpl<float,float>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_cm_kernel_t)hipsparseScsrmm)),err,ret);
            } else {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_cm_tmpl<hipFloatComplex,complex float>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_cm_kernel_t)hipsparseCcsrmm)),err,ret);
            }
        }
    } else {
        INFO_LOG("Calling row-major cuSparse CRS SpMMV");
        if (mat->traits.datatype & GHOST_DT_DOUBLE) {
            if (mat->traits.datatype & GHOST_DT_REAL) {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_rm_tmpl<double,double>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_rm_kernel_t)hipsparseDcsrmm2)),err,ret);
            } else {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_rm_tmpl<hipDoubleComplex,complex double>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_rm_kernel_t)hipsparseZcsrmm2)),err,ret);
            }
        } else {
            if (mat->traits.datatype & GHOST_DT_REAL) {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_rm_tmpl<float,float>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_rm_kernel_t)hipsparseScsrmm2)),err,ret);
            } else {
                GHOST_CALL_GOTO((ghost_cu_sell1spmmv_rm_tmpl<hipFloatComplex,complex float>(mat,lhs,rhs,traits,(cusparse_sell1_spmmv_rm_kernel_t)hipsparseCcsrmm2)),err,ret);
            }
        }
    }
    
    if (lhs != lhs_in) {
        lhs_in->fromVec(lhs_in,lhs,0,0);
        ghost_densemat_destroy(lhs);
    }
    if (rhs != rhs_in) {
        ghost_densemat_destroy(rhs);
    }

    if (mat->traits.datatype & GHOST_DT_DOUBLE) {
        if (mat->traits.datatype & GHOST_DT_REAL) {
            GHOST_CALL_GOTO((ghost_cu_sell1spmv_augfunc_tmpl<double,double>(lhs_in,rhs_in,traits)),err,ret);
        } else {
            GHOST_CALL_GOTO((ghost_cu_sell1spmv_augfunc_tmpl<hipDoubleComplex,complex double>(lhs_in,rhs_in,traits)),err,ret);
        }
    } else {
        if (mat->traits.datatype & GHOST_DT_REAL) {
            GHOST_CALL_GOTO((ghost_cu_sell1spmv_augfunc_tmpl<float,float>(lhs_in,rhs_in,traits)),err,ret);
        } else {
            GHOST_CALL_GOTO((ghost_cu_sell1spmv_augfunc_tmpl<hipFloatComplex,complex float>(lhs_in,rhs_in,traits)),err,ret);
        }
    }
    
    goto out;
err:
    ERROR_LOG("Error in SELL-1 SpMV!");
out:

    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);
    return ret;
}
