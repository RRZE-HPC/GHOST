#include "hip/hip_runtime.h"
#include "ghost/config.h"
#undef GHOST_HAVE_MPI
#include "ghost/types.h"
#include "ghost/crs.h"
#include "ghost/log.h"
#include "ghost/cu_util.h"
#include "ghost/cu_complex.h"
#include "ghost/util.h"

#include <complex.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

typedef hipsparseStatus_t (*crskernel_t) (hipsparseHandle_t handle, hipsparseOperation_t transA, 
        int m, int n, int nnz, const void           *alpha, 
        const hipsparseMatDescr_t descrA, 
        const void           *csrValA, 
        const void *csrRowPtrA, const void *csrColIndA,
        const void           *x, const void           *beta, 
        void           *y);

    template<typename dt1, typename dt2>
static ghost_error_t ghost_cu_crsspmv_tmpl(ghost_sparsemat_t *mat, ghost_densemat_t * lhs, ghost_densemat_t * rhs, ghost_spmv_flags_t options, va_list argp, crskernel_t crskernel)
{
    if (options & GHOST_SPMV_DOT_ANY) {
        ERROR_LOG("Localdot not implemented!");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    if (options & GHOST_SPMV_SHIFT) {
        ERROR_LOG("Shift not implemented!");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }

    hipsparseHandle_t cusparse_handle;
    hipsparseMatDescr_t descr;

    hipsparseCreateMatDescr(&descr);
    GHOST_CALL_RETURN(ghost_cu_cusparse_handle(&cusparse_handle));

    dt2 *localdot = NULL;
    dt1 *shift = NULL, scale, beta;

    one<dt1>(scale);

    GHOST_SPMV_PARSE_ARGS(options,argp,scale,beta,shift,localdot,dt2,dt1);

    if (options & GHOST_SPMV_AXPY) {
        one<dt1>(beta);
    } else {
        zero<dt1>(beta);
    }
    
    ghost_densemat_t *lhscompact, *rhscompact;
    void *lhsval, *rhsval;
    
    if (lhs->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) lhs before operation");
        GHOST_CALL_RETURN(lhs->clone(lhs,&lhscompact,lhs->traits.nrows,0,lhs->traits.ncols,0));
    } else {
        lhscompact = lhs;
    }
    if (rhs->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) v2 before operation");
        GHOST_CALL_RETURN(rhs->clone(rhs,&rhscompact,rhs->traits.nrows,0,rhs->traits.ncols,0));
    } else {
        rhscompact = rhs;
    }
    GHOST_CALL_RETURN(ghost_densemat_cu_valptr(lhscompact,&lhsval));
    GHOST_CALL_RETURN(ghost_densemat_cu_valptr(rhscompact,&rhsval));


    if (localdot || shift) {
        WARNING_LOG("Localdot or shift are not NULL, something went wrong!");
    } 

    crskernel(cusparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,mat->nrows,mat->ncols,mat->nnz,&scale,descr,(dt1 *)CR(mat)->cumat->val, CR(mat)->cumat->rpt, CR(mat)->cumat->col, (dt1 *)rhsval, &beta, (dt1 *)lhsval);

    if (lhscompact != lhs) {
        GHOST_CALL_RETURN(lhs->fromVec(lhs,lhscompact,0,0));
        lhscompact->destroy(lhscompact);
    }
    if (rhscompact != rhs) {
        rhscompact->destroy(rhscompact);
    }

    return GHOST_SUCCESS;

}

ghost_error_t ghost_cu_crs_spmv_selector(ghost_sparsemat_t *mat, ghost_densemat_t * lhs, ghost_densemat_t * rhs, ghost_spmv_flags_t options, va_list argp)
{
    if (mat->traits->datatype != lhs->traits.datatype) {
        ERROR_LOG("Mixed data types not implemented!");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }

    INFO_LOG("Calling cuSparse CRS SpMV");

    if (mat->traits->datatype & GHOST_DT_DOUBLE) {
        if (mat->traits->datatype & GHOST_DT_REAL) {
            return ghost_cu_crsspmv_tmpl<double,double>(mat,lhs,rhs,options,argp,(crskernel_t)hipsparseDcsrmv);
        } else {
            return ghost_cu_crsspmv_tmpl<hipDoubleComplex,complex double>(mat,lhs,rhs,options,argp,(crskernel_t)hipsparseZcsrmv);
        }
    } else {
        if (mat->traits->datatype & GHOST_DT_REAL) {
            return ghost_cu_crsspmv_tmpl<float,float>(mat,lhs,rhs,options,argp,(crskernel_t)hipsparseScsrmv);
        } else {
            return ghost_cu_crsspmv_tmpl<hipFloatComplex,complex float>(mat,lhs,rhs,options,argp,(crskernel_t)hipsparseCcsrmv);
        }
    }


    /*hipsparseHandle_t cusparse_handle;
      hipsparseMatDescr_t descr;

      hipsparseCreateMatDescr(&descr);
      GHOST_CALL_RETURN(ghost_cu_cusparse_handle(&cusparse_handle));

      double one = 1.;

      hipsparseDcsrmv(cusparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,mat->nrows,mat->ncols,mat->nnz,&one,descr,(const double *)CR(mat)->cumat->val, CR(mat)->cumat->rpt, CR(mat)->cumat->col, (const double *)rhs->cu_val, &one, (double *)lhs->cu_val);

    //ERROR_LOG("CUDA CRS spMV not implemented");
    //return GHOST_ERR_NOT_IMPLEMENTED;

    INFO_LOG("ready"); 
    return GHOST_SUCCESS;*/

}
