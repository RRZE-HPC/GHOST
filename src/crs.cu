#include "hip/hip_runtime.h"
#include "ghost/config.h"
#undef GHOST_HAVE_MPI
#include "ghost/types.h"
#include "ghost/crs.h"
#include "ghost/log.h"

#include <hip/hip_runtime.h>
#include <hipsparse.h>

ghost_error_t ghost_cu_crsspmv(ghost_sparsemat_t *mat, ghost_densemat_t * lhs, ghost_densemat_t * rhs, int options)
{
    ERROR_LOG("CUDA CRS spMV not implemented");
    return GHOST_ERR_NOT_IMPLEMENTED;

}
