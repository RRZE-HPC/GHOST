#include "hip/hip_runtime.h"
#define CUDAKERNEL
#include <ghost.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ghost_util.h>
#include <ghost_types.h>
#include <sell.h>
#include "ghost_complex.h"
#include <hip/hip_complex.h>

#define CHOOSE_KERNEL(dt1,dt2,ch, ...) \
	switch(ch) { \
		case 1: \
				SELL_kernel_CU_tmpl< dt1, dt2, 1 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		case 2: \
				SELL_kernel_CU_tmpl< dt1, dt2, 2 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		case 4: \
				SELL_kernel_CU_tmpl< dt1, dt2, 4 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		case 8: \
				SELL_kernel_CU_tmpl< dt1, dt2, 8 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		case 16: \
				 SELL_kernel_CU_tmpl< dt1, dt2, 16 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		case 32: \
				 SELL_kernel_CU_tmpl< dt1, dt2, 32 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		case 64: \
				 SELL_kernel_CU_tmpl< dt1, dt2, 64 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		case 256: \
				 SELL_kernel_CU_tmpl< dt1, dt2, 256 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		break; \
		default: \
				 DEBUG_LOG(2,"Calling ELLPACK kernel"); \
				 SELL_kernel_CU_ELLPACK_tmpl< dt1, dt2 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		}
	/*	default: \
				 return SELL_kernel_CU_ELLPACK_tmpl< dt1, dt2 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)ch),ch >>> ( __VA_ARGS__ ); \
		break; \
	}*/

template<typename T>
__device__ inline void zero(T &val)
{
	val = 0.;
}

template<>
__device__ inline void zero<hipFloatComplex>(hipFloatComplex &val)
{
	val = make_hipFloatComplex(0.,0.);
}

template<>
__device__ inline void zero<hipDoubleComplex>(hipDoubleComplex &val)
{
	val = make_hipDoubleComplex(0.,0.);
}

// val += val2*val3
template<typename T, typename T2>
__device__ inline T axpy(T val, T val2, T2 val3)
{
	return val+val2*val3;
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipFloatComplex>(hipFloatComplex val, hipFloatComplex val2, hipFloatComplex val3)
{
	return hipCaddf(val,hipCmulf(val2,val3));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,double>(hipFloatComplex val, hipFloatComplex val2, double val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,float>(hipFloatComplex val, hipFloatComplex val2, float val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex(val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipDoubleComplex>(hipFloatComplex val, hipFloatComplex val2, hipDoubleComplex val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)(hipCreal(val3)),(float)(hipCimag(val3)))));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,double>(hipDoubleComplex val, hipDoubleComplex val2, double val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex(val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,float>(hipDoubleComplex val, hipDoubleComplex val2, float val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipDoubleComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipDoubleComplex val3)
{
	return hipCadd(val,hipCmul(val2,val3));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipFloatComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipFloatComplex val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)(hipCrealf(val3)),(double)(hipCimagf(val3)))));
}

template<>
__device__ inline double axpy<double,hipFloatComplex>(double val, double val2, hipFloatComplex val3)
{
	return val+val2*(double)hipCrealf(val3);
}


template<>
__device__ inline double axpy<double,hipDoubleComplex>(double val, double val2, hipDoubleComplex val3)
{
	return val+val2*hipCreal(val3);
}

template<>
__device__ inline float axpy<float,hipFloatComplex>(float val, float val2, hipFloatComplex val3)
{
	return val+val2*hipCrealf(val3);
}


template<>
__device__ inline float axpy<float,hipDoubleComplex>(float val, float val2, hipDoubleComplex val3)
{
	return val+val2*(float)hipCreal(val3);
}

template<typename m_t, typename v_t>  
__global__ void SELL_kernel_CU_ELLPACK_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int j;
		v_t tmp;
		zero<v_t>(tmp);

		for (j=0; j<rowlen[i]; j++) {
			tmp = axpy<v_t,m_t>(tmp, rhs[col[i + j*nrowspadded]], val[i + j*nrowspadded]);
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
		else 
			lhs[i] = tmp;
	}
}

template<typename m_t, typename v_t, int chunkHeight>  
__global__ void SELL_kernel_CU_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

//	printf(">>> %d\n",nrows);
	if (i<nrows) {
		int cs, tid;
		if (chunkHeight == SELL_CUDA_BLOCKSIZE) {
		cs = chunkstart[blockIdx.x];
		tid = threadIdx.x;
		} else {
		cs = chunkstart[i/chunkHeight];
		tid = threadIdx.x%chunkHeight;
		}
		int j;
		v_t tmp;
		zero<v_t>(tmp);

		for (j=0; j<rowlen[i]; j++) {
//			printf("%d/%d: %f*%f\n",i,j,rhs[col[cs + tid + j*chunkHeight]], val[cs + tid + j*chunkHeight]);
			tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + tid + j*chunkHeight]], val[cs + tid + j*chunkHeight]);
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
		else 
			lhs[i] = tmp;
	}
}

/*template<typename m_t>  
__global__ void SELL_kernel_CU_cvec_tmpl(hipFloatComplex *lhs, hipFloatComplex *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int cs = chunkstart[blockIdx.x];
		int j;
		hipFloatComplex tmp = make_hipFloatComplex(0.,0.);


		for (j=0; j<rowlen[i]; j++) {
			tmp += make_hipFloatComplex(val[cs + threadIdx.x + j*SELL_LEN])  // TODO cast besser machen
				* rhs[col[cs + threadIdx.x + j*SELL_LEN]];
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] += tmp;
		else 
			lhs[i] = tmp;
	}
}*/

extern "C" void dd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void ds_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void dc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void dz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void sd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void ss_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void sc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void sz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

/*extern "C" void ds_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< double,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void dc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options){ return SELL_kernel_CU_tmpl< double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipFloatComplex*)lhs->CU_val,(hipFloatComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void dz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< double,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void sd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void ss_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void sc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,hipComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void sz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,hipComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,hipFloatComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }
*/
