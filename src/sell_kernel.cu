#include "hip/hip_runtime.h"
#include <ghost_config.h>
#undef GHOST_HAVE_MPI
#include <ghost_types.h>
#include <ghost_sell.h>
#include <ghost_complex.h>
#include <ghost_util.h>
#include <ghost_constants.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define SELL_CUDA_NBLOCKS (int)ceil(SELL(mat)->cumat->nrowsPadded/(double)(SELL_CUDA_THREADSPERBLOCK/SELL(mat)->T)) 

extern __shared__ char shared[];
extern int ghost_cu_device;

#define CHOOSE_KERNEL(dt1,dt2) {\
    if ((SELL(mat)->T > 32) || (SELL(mat)->T == 0) || (SELL(mat)->T & (SELL(mat)->T-1)))\
        WARNING_LOG("Invalid T: %d (must be power of two <33",SELL(mat)->T);\
    if (SELL(mat)->chunkHeight == SELL(mat)->nrowsPadded) {\
        if (SELL(mat)->T > 1) {\
            INFO_LOG("ELLPACK-T kernel not available. Switching to SELL-T kernel although we have only one chunk. Performance may suffer.");\
            size_t reqSmem = ghost_sizeofDataType(lhs->traits->datatype)*SELL_CUDA_THREADSPERBLOCK;\
            struct hipDeviceProp_t prop;\
            CU_safecall(hipGetDeviceProperties(&prop,ghost_cu_device));\
            if (prop.sharedMemPerBlock < reqSmem) {\
                WARNING_LOG("Not enough shared memory available! CUDA kernel will not execute!");\
            }\
            dim3 block(SELL_CUDA_THREADSPERBLOCK/SELL(mat)->T,SELL(mat)->T);\
            SELLT_kernel_CU_tmpl\
                <dt1,dt2>\
                <<< SELL_CUDA_NBLOCKS,block,reqSmem >>>\
                ((dt2 *)lhs->CU_val[0],(dt2 *)rhs->CU_val[0],options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLenPadded,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T);\
        } else {\
            SELL_kernel_CU_ELLPACK_tmpl\
                <dt1,dt2>\
                <<< SELL_CUDA_NBLOCKS,SELL_CUDA_THREADSPERBLOCK >>>\
                ((dt2 *)lhs->CU_val[0],(dt2 *)rhs->CU_val[0],options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T);\
        }\
    }else{\
        if (SELL(mat)->T > 1) {\
            size_t reqSmem = ghost_sizeofDataType(lhs->traits->datatype)*SELL_CUDA_THREADSPERBLOCK;\
            struct hipDeviceProp_t prop;\
            CU_safecall(hipGetDeviceProperties(&prop,ghost_cu_device));\
            if (prop.sharedMemPerBlock < reqSmem) {\
                WARNING_LOG("Not enough shared memory available! CUDA kernel will not execute!");\
            }\
            dim3 block(SELL_CUDA_THREADSPERBLOCK/SELL(mat)->T,SELL(mat)->T);\
            SELLT_kernel_CU_tmpl\
                <dt1,dt2>\
                <<< SELL_CUDA_NBLOCKS,block,reqSmem >>>\
                ((dt2 *)lhs->CU_val[0],(dt2 *)rhs->CU_val[0],options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLenPadded,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T);\
        } else {\
            SELL_kernel_CU_tmpl\
                <dt1,dt2>\
                <<< SELL_CUDA_NBLOCKS,SELL_CUDA_THREADSPERBLOCK >>>\
                ((dt2 *)lhs->CU_val[0],(dt2 *)rhs->CU_val[0],options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T);\
        }\
    }\
}

template<typename T>
__device__ inline void zero(T &val)
{
    val = 0.;
}

template<>
__device__ inline void zero<hipFloatComplex>(hipFloatComplex &val)
{
    val = make_hipFloatComplex(0.,0.);
}

template<>
__device__ inline void zero<hipDoubleComplex>(hipDoubleComplex &val)
{
    val = make_hipDoubleComplex(0.,0.);
}

// val += val2*val3
template<typename T, typename T2>
__device__ inline T axpy(T val, T val2, T2 val3)
{
    return val+val2*val3;
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipFloatComplex>(hipFloatComplex val, hipFloatComplex val2, hipFloatComplex val3)
{
    return hipCaddf(val,hipCmulf(val2,val3));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,double>(hipFloatComplex val, hipFloatComplex val2, double val3)
{
    return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,float>(hipFloatComplex val, hipFloatComplex val2, float val3)
{
    return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex(val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipDoubleComplex>(hipFloatComplex val, hipFloatComplex val2, hipDoubleComplex val3)
{
    return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)(hipCreal(val3)),(float)(hipCimag(val3)))));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,double>(hipDoubleComplex val, hipDoubleComplex val2, double val3)
{
    return hipCadd(val,hipCmul(val2,make_hipDoubleComplex(val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,float>(hipDoubleComplex val, hipDoubleComplex val2, float val3)
{
    return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipDoubleComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipDoubleComplex val3)
{
    return hipCadd(val,hipCmul(val2,val3));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipFloatComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipFloatComplex val3)
{
    return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)(hipCrealf(val3)),(double)(hipCimagf(val3)))));
}

template<>
__device__ inline double axpy<double,hipFloatComplex>(double val, double val2, hipFloatComplex val3)
{
    return val+val2*(double)hipCrealf(val3);
}


template<>
__device__ inline double axpy<double,hipDoubleComplex>(double val, double val2, hipDoubleComplex val3)
{
    return val+val2*hipCreal(val3);
}

template<>
__device__ inline float axpy<float,hipFloatComplex>(float val, float val2, hipFloatComplex val3)
{
    return val+val2*hipCrealf(val3);
}


template<>
__device__ inline float axpy<float,hipDoubleComplex>(float val, float val2, hipDoubleComplex val3)
{
    return val+val2*(float)hipCreal(val3);
}

template<typename m_t, typename v_t>  
__global__ void SELL_kernel_CU_ELLPACK_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen, int C, int T)
{
    UNUSED(C);
    UNUSED(T);

    int i = threadIdx.x+blockIdx.x*blockDim.x;

    if (i<nrows) {
        int j;
        v_t tmp;
        zero<v_t>(tmp);

        for (j=0; j<rowlen[i]; j++) {
            tmp = axpy<v_t,m_t>(tmp, rhs[col[i + j*nrowspadded]], val[i + j*nrowspadded]);
        }
        if (options & GHOST_SPMVM_AXPY)
            lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
        else 
            lhs[i] = tmp;
    }
}

template<typename m_t, typename v_t>  
__global__ void SELL_kernel_CU_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen, int C, int T)
{
    UNUSED(T);
    int i = threadIdx.x+blockIdx.x*blockDim.x;

    if (i<nrows) {
        int cs, tid;
        if (C == blockDim.x) {
            cs = chunkstart[blockIdx.x];
            tid = threadIdx.x;
        } else {
            cs = chunkstart[i/C];
            tid = threadIdx.x%C;
        }
        int j;
        v_t tmp;
        zero<v_t>(tmp);

        for (j=0; j<rowlen[i]; j++) {
            tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + tid + j*C]], val[cs + tid + j*C]);
        }
        if (options & GHOST_SPMVM_AXPY)
            lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
        else 
            lhs[i] = tmp;
    }
}

template<typename m_t, typename v_t>  
__global__ void SELLT_kernel_CU_tmpl(v_t *lhs, v_t *rhs, int options, ghost_midx_t nrows, ghost_midx_t nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen, ghost_midx_t C, int T)
{
    int i = threadIdx.x+blockIdx.x*blockDim.x;

    if (i<nrows) {
        int tib = threadIdx.x*blockDim.y+threadIdx.y;
        int cs, tid; // chunk start, thread row in block
        int j;
        v_t tmp;
        v_t *smem = (v_t *)shared;
        
        if (C == blockDim.x) {
            cs = chunkstart[blockIdx.x];
            tid = threadIdx.x;
        } else {
            cs = chunkstart[i/C];
            tid = threadIdx.x%C;
        }
        zero<v_t>(tmp);


        for (j=0; j<rowlen[i]/T; j++) {
            tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + tid + (threadIdx.y*rowlen[i]/T+j)*C]], val[cs + tid + (threadIdx.y*rowlen[i]/T+j)*C]);
        }
        smem[tib] = tmp;
        __syncthreads();
        
        if (T>2) {
            if (T>4) {
                if (T>8) {
                    if (T>16) {
                        if (threadIdx.y<16) {
                            smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+16],1.f);
                            __syncthreads();
                        }
                    }
                    if (threadIdx.y<8) {
                        smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+8],1.f);
                        __syncthreads();
                    }
                }
                if (threadIdx.y<4) {
                    smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+4],1.f);
                    __syncthreads();    
                }
            }
            if (threadIdx.y<2) {
                smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+2],1.f);
                __syncthreads();
            }
        }
        __syncthreads();
        
        if (threadIdx.y == 0) {
            if (options & GHOST_SPMVM_AXPY)
                lhs[i] = axpy<v_t,float>(lhs[i],axpy<v_t,float>(smem[tib],smem[tib+1],1.f),1.f);
            else 
                lhs[i] = axpy<v_t,float>(smem[tib],smem[tib+1],1.f);
        }
    }
}


extern "C" void dd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{
    CHOOSE_KERNEL(double,double);
}

extern "C" void ds_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(double,float);
}

extern "C" void dc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(double,hipFloatComplex);
}

extern "C" void dz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(double,hipDoubleComplex);
}

extern "C" void sd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(float,double);
}

extern "C" void ss_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(float,float);
}

extern "C" void sc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(float,hipFloatComplex);
}

extern "C" void sz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(float,hipDoubleComplex);
}

extern "C" void zd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,double);
}

extern "C" void zs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,float);
}

extern "C" void zc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,hipFloatComplex);
}

extern "C" void zz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,hipDoubleComplex);
}

extern "C" void cd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipFloatComplex,double);
}

extern "C" void cs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipFloatComplex,float);
}

extern "C" void cc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipFloatComplex,hipFloatComplex);
}

extern "C" void cz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ 
    CHOOSE_KERNEL(hipFloatComplex,hipDoubleComplex);
}
