#include "hip/hip_runtime.h"
#include <ghost_config.h>
#undef GHOST_HAVE_MPI
#include <ghost_types.h>
#include <ghost_sell.h>
#include <ghost_complex.h>
#include <ghost_util.h>
#include <ghost_constants.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

extern __shared__ char shared[];

//#define SWITCH_T

#define CHOOSE_KERNEL(func,dt1,dt2,ch, ...) \
	if (mat->T == 1) {\
		switch(ch) { \
			case 1: \
					func< dt1, dt2, 1 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			case 2: \
					func< dt1, dt2, 2 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			case 4: \
					func< dt1, dt2, 4 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			case 8: \
					func< dt1, dt2, 8 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			case 16: \
					 func< dt1, dt2, 16 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			case 32: \
					 func< dt1, dt2, 32 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			case 64: \
					 func< dt1, dt2, 64 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			case 256: \
					 func< dt1, dt2, 256 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
			break; \
			default: \
					 DEBUG_LOG(2,"Calling ELLPACK kernel"); \
					 SELL_kernel_CU_ELLPACK_tmpl< dt1, dt2 > <<< (int)ceil(SELL(mat)->cumat->nrows/(double)SELL_CUDA_BLOCKSIZE),SELL_CUDA_BLOCKSIZE >>> ( __VA_ARGS__ ); \
		}\
	}

template<typename T>
__device__ inline void zero(T &val)
{
	val = 0.;
}

template<>
__device__ inline void zero<hipFloatComplex>(hipFloatComplex &val)
{
	val = make_hipFloatComplex(0.,0.);
}

template<>
__device__ inline void zero<hipDoubleComplex>(hipDoubleComplex &val)
{
	val = make_hipDoubleComplex(0.,0.);
}

// val += val2*val3
template<typename T, typename T2>
__device__ inline T axpy(T val, T val2, T2 val3)
{
	return val+val2*val3;
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipFloatComplex>(hipFloatComplex val, hipFloatComplex val2, hipFloatComplex val3)
{
	return hipCaddf(val,hipCmulf(val2,val3));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,double>(hipFloatComplex val, hipFloatComplex val2, double val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,float>(hipFloatComplex val, hipFloatComplex val2, float val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex(val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipDoubleComplex>(hipFloatComplex val, hipFloatComplex val2, hipDoubleComplex val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)(hipCreal(val3)),(float)(hipCimag(val3)))));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,double>(hipDoubleComplex val, hipDoubleComplex val2, double val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex(val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,float>(hipDoubleComplex val, hipDoubleComplex val2, float val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipDoubleComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipDoubleComplex val3)
{
	return hipCadd(val,hipCmul(val2,val3));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipFloatComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipFloatComplex val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)(hipCrealf(val3)),(double)(hipCimagf(val3)))));
}

template<>
__device__ inline double axpy<double,hipFloatComplex>(double val, double val2, hipFloatComplex val3)
{
	return val+val2*(double)hipCrealf(val3);
}


template<>
__device__ inline double axpy<double,hipDoubleComplex>(double val, double val2, hipDoubleComplex val3)
{
	return val+val2*hipCreal(val3);
}

template<>
__device__ inline float axpy<float,hipFloatComplex>(float val, float val2, hipFloatComplex val3)
{
	return val+val2*hipCrealf(val3);
}


template<>
__device__ inline float axpy<float,hipDoubleComplex>(float val, float val2, hipDoubleComplex val3)
{
	return val+val2*(float)hipCreal(val3);
}

template<typename m_t, typename v_t>  
__global__ void SELL_kernel_CU_ELLPACK_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int j;
		v_t tmp;
		zero<v_t>(tmp);

		for (j=0; j<rowlen[i]; j++) {
			tmp = axpy<v_t,m_t>(tmp, rhs[col[i + j*nrowspadded]], val[i + j*nrowspadded]);
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
		else 
			lhs[i] = tmp;
	}
}

template<typename m_t, typename v_t, int chunkHeight>  
__global__ void SELL_kernel_CU_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int cs, tid;
		if (chunkHeight == SELL_CUDA_BLOCKSIZE) {
			cs = chunkstart[blockIdx.x];
			tid = threadIdx.x;
		} else {
			cs = chunkstart[i/chunkHeight];
			tid = threadIdx.x%chunkHeight;
		}
		int j;
		v_t tmp;
		zero<v_t>(tmp);

		for (j=0; j<rowlen[i]; j++) {
			tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + tid + j*chunkHeight]], val[cs + tid + j*chunkHeight]);
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
		else 
			lhs[i] = tmp;
	}
}

template<typename m_t, typename v_t, int chunkHeight, int T>  
__global__ void SELLT_kernel_CU_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int tib = threadIdx.x*blockDim.x+threadIdx.y; // thread idx in block
		int cs, tid; // chunk start, thread row in block
		int j;
		v_t tmp;
		v_t *smem = (v_t *)smem;
		
		if (chunkHeight == SELL_CUDA_BLOCKSIZE) {
			cs = chunkstart[blockIdx.x];
			tid = threadIdx.x;
		} else {
			cs = chunkstart[i/chunkHeight];
			tid = threadIdx.x%chunkHeight;
		}
		zero<v_t>(tmp);

		for (j=0; j<rowlen[i]; j++) {
			tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + tid + T*j*chunkHeight]], val[cs + tid + T*j*chunkHeight]);
		}
		smem[threadIdx.x*T+threadIdx.y] = tmp;
		if (T>2) {
			if (T>4) {
				if (T>8) {
					if (T>16) {
						if (threadIdx.y<16)
							smem[threadIdx.x*T]+=smem[threadIdx.x*T+16];
					}
					if (threadIdx.y<8)
						smem[threadIdx.x*T]+=smem[threadIdx.x*T+8];
				}
				if (threadIdx.y<4)
					smem[threadIdx.x*T]+=smem[threadIdx.x*T+4];
			}
			if (threadIdx.y<2)
				smem[threadIdx.x*T]+=smem[threadIdx.x*T+2];
		}
		
		if (threadIdx.y == 0) {
			if (options & GHOST_SPMVM_AXPY)
				lhs[i] = axpy<v_t,float>(lhs[i],smem[threadIdx.x*T]+smem[threadIdx.x*T+1],1.f);
			else 
				lhs[i] = tmp;
		}
	}
}

/*template<typename m_t>  
__global__ void SELL_kernel_CU_cvec_tmpl(hipFloatComplex *lhs, hipFloatComplex *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int cs = chunkstart[blockIdx.x];
		int j;
		hipFloatComplex tmp = make_hipFloatComplex(0.,0.);


		for (j=0; j<rowlen[i]; j++) {
			tmp += make_hipFloatComplex(val[cs + threadIdx.x + j*SELL_LEN])  // TODO cast besser machen
				* rhs[col[cs + threadIdx.x + j*SELL_LEN]];
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] += tmp;
		else 
			lhs[i] = tmp;
	}
}*/

extern "C" void dd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void ds_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void dc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void dz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(double,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void sd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void ss_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void sc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void sz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(float,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void cz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipFloatComplex,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipFloatComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,double,SELL(mat)->chunkHeight,(double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,float,SELL(mat)->chunkHeight,(float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,hipFloatComplex,SELL(mat)->chunkHeight,(hipFloatComplex *)lhs->CU_val,(hipFloatComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

extern "C" void zz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ CHOOSE_KERNEL(hipDoubleComplex,hipDoubleComplex,SELL(mat)->chunkHeight,(hipDoubleComplex *)lhs->CU_val,(hipDoubleComplex *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen) }

/*extern "C" void ds_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< double,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void dc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options){ return SELL_kernel_CU_tmpl< double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipFloatComplex*)lhs->CU_val,(hipFloatComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void dz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< double,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(double *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void sd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void ss_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void sc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,hipComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void sz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< float,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(float *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,hipComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void cz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipFloatComplex,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zd_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,double > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zs_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,float > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zc_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,hipFloatComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }

extern "C" void zz_SELL_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return SELL_kernel_CU_tmpl< hipDoubleComplex,hipDoubleComplex > <<<(int)ceil(SELL(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,SELL(mat)->cumat->nrows,SELL(mat)->cumat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(hipDoubleComplex*)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen); }
*/
