#include "hip/hip_runtime.h"
#include "ghost/config.h"
#undef GHOST_HAVE_MPI
#undef GHOST_HAVE_INSTR_LIKWID
#include "ghost/types.h"
#include "ghost/sell.h"
#include "ghost/complex.h"
#include "ghost/instr.h"
#include "ghost/log.h"
#include "ghost/error.h"
#include "ghost/util.h"
#include "ghost/math.h"

#include <hip/hip_complex.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <complex.h>

#include "ghost/cu_complex.h"

#define MAX_COLS_PER_BLOCK 16
#define SELL_CUDA_NBLOCKS (int)ceil(mat->nrowsPadded/ceil((double)(SELL_CUDA_THREADSPERBLOCK/((double)SELL(mat)->T*(double)(MIN(rhs->traits.ncols,MAX_COLS_PER_BLOCK))))))
//#define SELLT_STRIDE_ONE
#define LOCALDOT_ONTHEFLY

extern __shared__ char shared[];

#define CALL(func,dt1,dt2,dt2_host,b1,b2,b3,b4,b5,...){\
    dt2 shift, scale, beta;\
    GHOST_SPMV_PARSE_ARGS(flags,argp,scale,beta,shift,localdot,dt2_host,dt2);\
    func<dt1,dt2,b1,b2,b3,b4,b5><<<__VA_ARGS__>>>((dt2 *)lhs->cu_val,(dt2 *)rhs->cu_val,flags,mat->nrows,mat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,shift,scale,beta,(dt2 *)cu_localdot);\
}\

#define SWITCH_BOOLS(func,dt1,dt2,dt2_host,...)\
    if (flags & GHOST_SPMV_AXPY) {\
        if (flags & GHOST_SPMV_AXPBY) {\
            if (flags & GHOST_SPMV_SCALE) {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,true,true,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,true,true,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,true,true,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,true,true,false,false,__VA_ARGS__)\
                    }\
                }\
            } else {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,true,false,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,true,false,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,true,false,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,true,false,false,false,__VA_ARGS__)\
                    }\
                }\
            }\
        } else {\
            if (flags & GHOST_SPMV_SCALE) {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,false,true,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,false,true,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,false,true,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,false,true,false,false,__VA_ARGS__)\
                    }\
                }\
            } else {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,false,false,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,false,false,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,true,false,false,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,true,false,false,false,false,__VA_ARGS__)\
                    }\
                }\
            }\
        }\
    } else {\
        if (flags & GHOST_SPMV_AXPBY) {\
            if (flags & GHOST_SPMV_SCALE) {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,true,true,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,true,true,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,true,true,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,true,true,false,false,__VA_ARGS__)\
                    }\
                }\
            } else {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,true,false,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,true,false,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,true,false,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,true,false,false,false,__VA_ARGS__)\
                    }\
                }\
            }\
        } else {\
            if (flags & GHOST_SPMV_SCALE) {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,false,true,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,false,true,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,false,true,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,false,true,false,false,__VA_ARGS__)\
                    }\
                }\
            } else {\
                if (flags & GHOST_SPMV_SHIFT) {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,false,false,true,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,false,false,true,false,__VA_ARGS__)\
                    }\
                } else {\
                    if (flags & GHOST_SPMV_DOT) {\
                        CALL(func,dt1,dt2,dt2_host,false,false,false,false,true,__VA_ARGS__)\
                    } else {\
                        CALL(func,dt1,dt2,dt2_host,false,false,false,false,false,__VA_ARGS__)\
                    }\
                }\
            }\
        }\
    }\


#ifdef LOCALDOT_ONTHEFLY
#define PROCESS_LOCALDOT(dt2_host)\
        GHOST_INSTR_START(spmv_cuda_dot_reduction)\
        int block, col;\
        INFO_LOG("Experimental local dot product with final reduction over %d blocks!",SELL_CUDA_NBLOCKS);\
        dt2_host *localdot_blocks;\
        GHOST_CALL_RETURN(ghost_malloc((void **)&localdot_blocks,sizeof(dt2_host)*rhs->traits.ncols*3*SELL_CUDA_NBLOCKS));\
        GHOST_CALL_RETURN(ghost_cu_download(localdot_blocks,cu_localdot,sizeof(dt2_host)*rhs->traits.ncols*3*SELL_CUDA_NBLOCKS));\
        _Pragma("omp parallel for private(block)")\
        for (col=0; col<rhs->traits.ncols; col++) {\
            localdot[col                      ] = 0;\
            localdot[col + 1*rhs->traits.ncols] = 0;\
            localdot[col + 2*rhs->traits.ncols] = 0;\
            for (block=0; block<SELL_CUDA_NBLOCKS; block++) {\
                localdot[col                      ] += localdot_blocks[                                        col*SELL_CUDA_NBLOCKS + block];\
                localdot[col + 1*rhs->traits.ncols] += localdot_blocks[1*SELL_CUDA_NBLOCKS*rhs->traits.ncols + col*SELL_CUDA_NBLOCKS + block];\
                localdot[col + 2*rhs->traits.ncols] += localdot_blocks[2*SELL_CUDA_NBLOCKS*rhs->traits.ncols + col*SELL_CUDA_NBLOCKS + block];\
            }\
        }\
        free(localdot_blocks);\
        GHOST_INSTR_STOP(spmv_cuda_dot_reduction)
#else
#define PROCESS_LOCALDOT(dt2_host)\
        GHOST_INSTR_START(spmv_cuda_dot)\
          INFO_LOG("Not doing the local dot product on-the-fly!");\
          memset(localdot,0,rhs->traits.ncols*3*sizeof(dt2_host));\
          lhs->dot(lhs,&localdot[0],lhs);\
          lhs->dot(lhs,&localdot[rhs->traits.ncols],rhs);\
          lhs->dot(rhs,&localdot[2*rhs->traits.ncols],rhs);\
          GHOST_INSTR_STOP(spmv_cuda_dot)
#endif

#define CHOOSE_KERNEL(dt1,dt2,dt2_host) {\
    ghost_error_t ret = GHOST_SUCCESS;\
    int cu_device;\
    GHOST_CALL_RETURN(ghost_cu_device(&cu_device));\
    dt2 *cu_localdot = NULL;\
    dt2 *cu_shift = NULL;\
    dt2_host *localdot = NULL;\
    dt2 *shift, scale, beta;\
    GHOST_SPMV_PARSE_ARGS(flags,argp,scale,beta,shift,localdot,dt2_host,dt2);\
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cu_localdot,sizeof(dt2)*rhs->traits.ncols*3*SELL_CUDA_NBLOCKS));\
    size_t shiftsize = sizeof(dt2)*(flags & GHOST_SPMV_SHIFT?1:(flags & GHOST_SPMV_VSHIFT?rhs->traits.ncols:0));\
    GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cu_shift,shiftsize));\
    GHOST_CALL_RETURN(ghost_cu_upload(cu_shift,shift,shiftsize));\
    if ((SELL(mat)->T > 128) || (SELL(mat)->T == 0) || (SELL(mat)->T & (SELL(mat)->T-1)))\
    WARNING_LOG("Invalid T: %d (must be power of two and T <= 128)",SELL(mat)->T);\
    struct hipDeviceProp_t prop;\
    CUDA_CALL_RETURN(hipGetDeviceProperties(&prop,cu_device));\
    GHOST_INSTR_START(spmv_cuda)\
    if (rhs->traits.flags & (GHOST_DENSEMAT_VIEW | GHOST_DENSEMAT_SCATTERED)) {\
        if (!ghost_bitmap_iscompact(rhs->ldmask)) {\
            ERROR_LOG("CUDA SpMV with masked out rows not yet implemented");\
            return GHOST_ERR_NOT_IMPLEMENTED;\
        }\
        if (!ghost_bitmap_isequal(rhs->trmask,lhs->trmask) || !ghost_bitmap_isequal(rhs->ldmask,lhs->ldmask)) {\
            ERROR_LOG("CUDA SpMV with differently masked densemats not yet implemented");\
            return GHOST_ERR_NOT_IMPLEMENTED;\
        }\
        char colfield[rhs->traits.ncolsorig];\
        char rowfield[rhs->traits.nrowsorig];\
        char *cucolfield, *curowfield;\
        ghost_densemat_mask2charfield(rhs->trmask,rhs->traits.ncolsorig,colfield);\
        ghost_densemat_mask2charfield(rhs->ldmask,rhs->traits.nrowsorig,rowfield);\
        GHOST_CALL_RETURN(ghost_cu_malloc((void **)&cucolfield,rhs->traits.ncolsorig));\
        GHOST_CALL_RETURN(ghost_cu_malloc((void **)&curowfield,rhs->traits.nrowsorig));\
        GHOST_CALL_RETURN(ghost_cu_upload(cucolfield,colfield,rhs->traits.ncolsorig));\
        GHOST_CALL_RETURN(ghost_cu_upload(curowfield,rowfield,rhs->traits.nrowsorig));\
        if (SELL(mat)->T > 1) {\
            WARNING_LOG("SELL-T kernel for multiple vectors not implemented, falling back to SELL-1!");\
        }\
        int blockheight = PAD((int)ceil((double)SELL_CUDA_THREADSPERBLOCK/rhs->traits.ncols),SELL(mat)->chunkHeight);\
        if (blockheight*rhs->traits.ncols > 1024) {\
            WARNING_LOG("Too many threads! (FIXME)");\
        }\
        dim3 block(blockheight,rhs->traits.ncols);\
        SELL_kernel_scattered_CU_tmpl<dt1,dt2><<<(int)ceil(mat->nrowsPadded/(double)blockheight),block>>>((dt2 *)lhs->cu_val,lhs->traits.nrowspadded,(dt2 *)rhs->cu_val,rhs->traits.nrowspadded,flags,rhs->traits.nrowsorig,mat->nrowsPadded,rhs->traits.ncolsorig,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,cucolfield,curowfield,cu_shift,scale,beta,cu_localdot,flags&GHOST_SPMV_AXPY,flags&GHOST_SPMV_AXPBY,flags&GHOST_SPMV_SCALE,flags&GHOST_SPMV_SHIFT,flags&GHOST_SPMV_VSHIFT,flags&GHOST_SPMV_DOT);\
    } else {\
        if (rhs->traits.ncols > 1) {\
            if (SELL(mat)->T > 1) {\
                WARNING_LOG("SELL-T kernel for multiple vectors nor implemented, falling back to SELL-1!");\
            }\
            int blockheight = PAD((int)ceil((double)SELL_CUDA_THREADSPERBLOCK/MIN(rhs->traits.ncols,MAX_COLS_PER_BLOCK)),SELL(mat)->chunkHeight);\
            if (blockheight*MIN(rhs->traits.ncols,MAX_COLS_PER_BLOCK) > 1024) {\
                WARNING_LOG("Too many threads! (FIXME)");\
            }\
            if (rhs->traits.ncols > MAX_COLS_PER_BLOCK) {\
                WARNING_LOG("Will have a loop over the vectors!");\
            }\
            dim3 block(blockheight,MIN(rhs->traits.ncols,MAX_COLS_PER_BLOCK));\
            size_t reqSmem = 0;\
            if (flags & GHOST_SPMV_DOT) {\
                reqSmem = sizeof(dt2)*32*block.y;\
            }\
            if (prop.sharedMemPerBlock < reqSmem) {\
                WARNING_LOG("Not enough shared memory available! CUDA kernel will not execute!");\
            }\
            INFO_LOG("grid %d block %dx%d shmem %zu",(int)ceil(mat->nrowsPadded/(double)blockheight),block.x,block.y,reqSmem);\
            if (rhs->traits.storage == GHOST_DENSEMAT_COLMAJOR) {\
                SELL_kernel_CU_tmpl<dt1,dt2><<<(int)ceil(mat->nrowsPadded/(double)blockheight),block,reqSmem>>>((dt2 *)lhs->cu_val,lhs->traits.nrowspadded,(dt2 *)rhs->cu_val,rhs->traits.nrowspadded,flags,mat->nrows,mat->nrowsPadded,rhs->traits.ncols/block.y,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,cu_shift,scale,beta,cu_localdot,flags&GHOST_SPMV_AXPY,flags&GHOST_SPMV_AXPBY,flags&GHOST_SPMV_SCALE,flags&GHOST_SPMV_SHIFT,flags&GHOST_SPMV_VSHIFT,flags&GHOST_SPMV_DOT);\
            } else {\
                INFO_LOG("Experimental row-major CUDA SELL-SpMMV");\
                dim3 newblock(32,32);\
                SELL_kernel_CU_rm_tmpl<dt1,dt2><<<(int)ceil(mat->nrowsPadded/(double)32),newblock,reqSmem>>>((dt2 *)lhs->cu_val,lhs->traits.ncolspadded,(dt2 *)rhs->cu_val,rhs->traits.ncolspadded,flags,mat->nrows,mat->nrowsPadded,rhs->traits.ncols/newblock.x,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,cu_shift,scale,beta,cu_localdot,flags&GHOST_SPMV_AXPY,flags&GHOST_SPMV_AXPBY,flags&GHOST_SPMV_SCALE,flags&GHOST_SPMV_SHIFT,flags&GHOST_SPMV_VSHIFT,flags&GHOST_SPMV_DOT);\
            }\
        } else {\
            if (SELL(mat)->chunkHeight == mat->nrowsPadded) {\
                if (SELL(mat)->T > 1) {\
                    INFO_LOG("ELLPACK-T kernel not available. Switching to SELL-T kernel although we have only one chunk. Performance may suffer.");\
                    size_t reqSmem;\
                    ghost_datatype_size(&reqSmem,lhs->traits.datatype);\
                    reqSmem *= SELL_CUDA_THREADSPERBLOCK;\
                    if (prop.sharedMemPerBlock < reqSmem) {\
                        WARNING_LOG("Not enough shared memory available! CUDA kernel will not execute!");\
                    }\
                    dim3 block(SELL_CUDA_THREADSPERBLOCK/SELL(mat)->T,SELL(mat)->T);\
                    SELLT_kernel_CU_tmpl<dt1,dt2><<<SELL_CUDA_NBLOCKS,block,reqSmem>>>((dt2 *)lhs->cu_val,(dt2 *)rhs->cu_val,flags,mat->nrows,mat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,cu_shift,scale,beta,cu_localdot,flags&GHOST_SPMV_AXPY,flags&GHOST_SPMV_AXPBY,flags&GHOST_SPMV_SCALE,flags&GHOST_SPMV_SHIFT,flags&GHOST_SPMV_VSHIFT,flags&GHOST_SPMV_DOT);\
                    /*SWITCH_BOOLS(SELLT_kernel_CU_tmpl,dt1,dt2,SELL_CUDA_NBLOCKS,block,reqSmem)*/\
                } else {\
                    SELL_kernel_CU_ELLPACK_tmpl<dt1,dt2><<<SELL_CUDA_NBLOCKS,SELL_CUDA_THREADSPERBLOCK>>>((dt2 *)lhs->cu_val,lhs->traits.nrowspadded,(dt2 *)rhs->cu_val,rhs->traits.nrowspadded,flags,mat->nrows,mat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,cu_shift,scale,beta,cu_localdot,flags&GHOST_SPMV_AXPY,flags&GHOST_SPMV_AXPBY,flags&GHOST_SPMV_SCALE,flags&GHOST_SPMV_SHIFT,flags&GHOST_SPMV_VSHIFT,flags&GHOST_SPMV_DOT);\
                    /*SWITCH_BOOLS(SELL_kernel_CU_ELLPACK_tmpl,dt1,dt2,SELL_CUDA_NBLOCKS,SELL_CUDA_THREADSPERBLOCK)*/\
                }\
            }else{\
                if (SELL(mat)->T > 1) {\
                    size_t reqSmem;\
                    ghost_datatype_size(&reqSmem,lhs->traits.datatype);\
                    reqSmem *= SELL_CUDA_THREADSPERBLOCK;\
                    struct hipDeviceProp_t prop;\
                    CUDA_CALL_RETURN(hipGetDeviceProperties(&prop,cu_device));\
                    if (prop.sharedMemPerBlock < reqSmem) {\
                        WARNING_LOG("Not enough shared memory available! CUDA kernel will not execute!");\
                    }\
                    dim3 block(SELL_CUDA_THREADSPERBLOCK/SELL(mat)->T,SELL(mat)->T);\
                    SELLT_kernel_CU_tmpl<dt1,dt2><<<SELL_CUDA_NBLOCKS,block,reqSmem>>>((dt2 *)lhs->cu_val,(dt2 *)rhs->cu_val,flags,mat->nrows,mat->nrowsPadded,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,cu_shift,scale,beta,cu_localdot,flags&GHOST_SPMV_AXPY,flags&GHOST_SPMV_AXPBY,flags&GHOST_SPMV_SCALE,flags&GHOST_SPMV_SHIFT,flags&GHOST_SPMV_VSHIFT,flags&GHOST_SPMV_DOT);\
                    /*SWITCH_BOOLS(SELLT_kernel_CU_tmpl,dt1,dt2,SELL_CUDA_NBLOCKS,block,reqSmem)*/\
                } else {\
                    int blockheight = PAD((int)ceil((double)SELL_CUDA_THREADSPERBLOCK/rhs->traits.ncols),SELL(mat)->chunkHeight);\
                    if (blockheight*rhs->traits.ncols > 1024) {\
                        WARNING_LOG("Too many threads! (FIXME)");\
                    }\
                    size_t reqSmem;\
                    ghost_datatype_size(&reqSmem,lhs->traits.datatype);\
                    reqSmem *= blockheight*rhs->traits.ncols;\
                    if (prop.sharedMemPerBlock < reqSmem) {\
                        WARNING_LOG("Not enough shared memory available! CUDA kernel will not execute!");\
                    }\
                    dim3 block(blockheight,rhs->traits.ncols);\
                    SELL_kernel_CU_tmpl<dt1,dt2><<<(int)ceil(mat->nrowsPadded/(double)blockheight),block,reqSmem>>>((dt2 *)lhs->cu_val,lhs->traits.nrowspadded,(dt2 *)rhs->cu_val,rhs->traits.nrowspadded,flags,mat->nrows,mat->nrowsPadded,rhs->traits.ncols,SELL(mat)->cumat->rowLen,SELL(mat)->cumat->col,(dt1 *)SELL(mat)->cumat->val,SELL(mat)->cumat->chunkStart,SELL(mat)->cumat->chunkLen,SELL(mat)->chunkHeight,SELL(mat)->T,cu_shift,scale,beta,cu_localdot,flags&GHOST_SPMV_AXPY,flags&GHOST_SPMV_AXPBY,flags&GHOST_SPMV_SCALE,flags&GHOST_SPMV_SHIFT,flags&GHOST_SPMV_VSHIFT,flags&GHOST_SPMV_DOT);\
                    /*SWITCH_BOOLS(SELL_kernel_CU_tmpl,dt1,dt2,SELL_CUDA_NBLOCKS,SELL_CUDA_THREADSPERBLOCK)*/\
                }\
            }\
        }\
    }\
    hipDeviceSynchronize();\
    GHOST_INSTR_STOP(spmv_cuda)\
    if (flags & GHOST_SPMV_DOT) {\
        PROCESS_LOCALDOT(dt2_host)\
    }\
    GHOST_CALL_RETURN(ghost_cu_free(cu_localdot));\
    GHOST_CALL_RETURN(ghost_cu_free(cu_shift));\
    return ret;\
}

template<typename v_t>
__device__ inline
v_t shfl_down(v_t var, unsigned int srcLane) {
    return __shfl_down(var, srcLane, warpSize);
}

template<>
__device__ inline
double shfl_down<double>(double var, unsigned int srcLane) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_down(a.x, srcLane, warpSize);
    a.y = __shfl_down(a.y, srcLane, warpSize);
    return *reinterpret_cast<double*>(&a);
}

template<>
__device__ inline
hipFloatComplex shfl_down<hipFloatComplex>(hipFloatComplex var, unsigned int srcLane) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_down(a.x, srcLane, warpSize);
    a.y = __shfl_down(a.y, srcLane, warpSize);
    return *reinterpret_cast<hipFloatComplex*>(&a);
}

template<>
__device__ inline
hipDoubleComplex shfl_down<hipDoubleComplex>(hipDoubleComplex var, unsigned int srcLane) {
    int4 a = *reinterpret_cast<int4*>(&var);
    a.x = __shfl_down(a.x, srcLane, warpSize);
    a.y = __shfl_down(a.y, srcLane, warpSize);
    a.z = __shfl_down(a.z, srcLane, warpSize);
    a.w = __shfl_down(a.w, srcLane, warpSize);
    return *reinterpret_cast<hipDoubleComplex*>(&a);
}

template<typename v_t>
__inline__ __device__
v_t warpReduceSum(v_t val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2) { 
        val = axpy<v_t>(val,shfl_down(val, offset),1.f);
    }
    return val;
}

template<typename v_t>
__inline__ __device__
v_t blockReduceSum(v_t val) {

    v_t * shmem = (v_t *)shared; // Shared mem for 32 partial sums

    int lane = (threadIdx.x % warpSize) + (32*threadIdx.y);
    int wid = (threadIdx.x / warpSize) + (32*threadIdx.y);

    val = warpReduceSum(val);     // Each warp performs partial reduction

    if (threadIdx.x%warpSize == 0) shmem[wid]=val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    if (threadIdx.x < blockDim.x / warpSize) {
        val = shmem[lane];
    } else {
        zero<v_t>(val);
    }

    if (threadIdx.x/warpSize == 0) val = warpReduceSum(val); //Final reduce within first warp

    return val;
}

template<typename v_t>
__global__ void deviceReduceKernel(v_t *in, v_t* out, int N) {
    v_t sum;
    zero<v_t>(sum);
    //reduce multiple elements per thread
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
            i < N; 
            i += blockDim.x * gridDim.x) {
        sum = axpy<v_t>(sum,in[i],1.f);
    }
    sum = blockReduceSum(sum);
    if (threadIdx.x==0)
        out[blockIdx.x]=sum;
}

    template<typename m_t, typename v_t>
__global__ void SELL_kernel_CU_ELLPACK_tmpl(v_t *lhs, int lhs_lda, v_t *rhs, int rhs_lda, ghost_spmv_flags_t flags, int nrows, int nrowspadded, ghost_lidx_t *rowlen, ghost_lidx_t *col, m_t *val, ghost_lidx_t *chunkstart, ghost_lidx_t *chunklen, int C, int T, v_t *shift, v_t alpha, v_t beta, v_t *localdot, const bool do_axpy, const bool do_axpby, const bool do_scale, const bool do_shift, const bool do_vshift, const bool do_localdot)
{
    UNUSED(C);
    UNUSED(T);

    int i = threadIdx.x+blockIdx.x*blockDim.x;

    if (i<nrows) {
        int j;
        v_t tmp;
        zero<v_t>(tmp);

        for (j=0; j<rowlen[i]; j++) {
            tmp = axpy<v_t,m_t>(tmp, rhs[col[i + j*nrowspadded]], val[i + j*nrowspadded]);
        }

        if (do_shift) {
            tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*threadIdx.y+i],scale2<v_t,float>(shift[0],-1.f));
        }
        if (do_vshift) {
            tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*threadIdx.y+i],scale2<v_t,float>(shift[threadIdx.y],-1.f));
        }
        if (do_scale) {
            tmp = scale<v_t>(alpha,tmp);
        }
        if (do_axpy) {
            lhs[lhs_lda*threadIdx.y+i] = axpy<v_t,float>(lhs[lhs_lda*threadIdx.y+i],tmp,1.f);
        } else if (do_axpby) {
            lhs[lhs_lda*threadIdx.y+i] = axpy<v_t,float>(scale<v_t>(lhs[lhs_lda*threadIdx.y+i],beta),tmp,1.f);
        } else {
            lhs[lhs_lda*threadIdx.y+i] = tmp;
        }
    }
}

    template<typename m_t, typename v_t>  
__global__ void SELL_kernel_scattered_CU_tmpl(v_t *lhs, int lhs_lda, v_t *rhs, int rhs_lda, ghost_spmv_flags_t flags, int nrowsorig, int nrowspadded, int ncolsorig, ghost_lidx_t *rowlen, ghost_lidx_t *col, m_t *val, ghost_lidx_t *chunkstart, ghost_lidx_t *chunklen, int C, int T, char *colmask, char *rowmask, v_t *shift, v_t alpha, v_t beta, v_t *localdot, const bool do_axpy, const bool do_axpby, const bool do_scale, const bool do_shift, const bool do_vshift, const bool do_localdot)
{
    UNUSED(T);
    int i = threadIdx.x+blockIdx.x*blockDim.x;

    if (i<nrowsorig) {
        int c = 0;
        int set = 0;
        for (c=0; c<ncolsorig; c++) {
            if (colmask[c]) {
                if (set == threadIdx.y) {
                    break;
                }
                set++;
            }
        }
        int cs, tid;
        if (C == blockDim.x) {
            cs = chunkstart[blockIdx.x];
            tid = threadIdx.x;
        } else {
            cs = chunkstart[i/C];
            tid = threadIdx.x%C;
        }
        int j;
        v_t tmp;

        zero<v_t>(tmp);

        for (j=0; j<rowlen[i]; j++) {
            tmp = axpy<v_t,m_t>(tmp, rhs[rhs_lda*c+col[cs + tid + j*C]], val[cs+tid+j*C]);
        }

        if (do_shift) {
            tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*c+i],scale2<v_t,float>(shift[0],-1.f));
        }
        if (do_vshift) {
            tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*c+i],scale2<v_t,float>(shift[c],-1.f));
        }
        if (do_scale) {
            tmp = scale<v_t>(alpha,tmp);
        }
        if (do_axpy) {
            lhs[lhs_lda*c+i] = axpy<v_t,float>(lhs[lhs_lda*c+i],tmp,1.f);
        } else if (do_axpby) {
            lhs[lhs_lda*c+i] = axpy<v_t,float>(scale<v_t>(lhs[lhs_lda*c+i],beta),tmp,1.f);
        } else {
            lhs[lhs_lda*c+i] = tmp;
        }
    }
}

    template<typename m_t, typename v_t>  
__global__ void SELL_kernel_CU_rm_tmpl(v_t *lhs, int lhs_lda, v_t *rhs, int rhs_lda, ghost_spmv_flags_t flags, int nrows, int nrowspadded, int ncols, ghost_lidx_t *rowlen, ghost_lidx_t *mcol, m_t *val, ghost_lidx_t *chunkstart, ghost_lidx_t *chunklen, int C, int T, v_t *shift, v_t alpha, v_t beta, v_t *localdot, const bool do_axpy, const bool do_axpby, const bool do_scale, const bool do_shift, const bool do_vshift, const bool do_localdot)
{
    UNUSED(T);
    int i = threadIdx.y+blockIdx.x*blockDim.y;
    int colblock,col;

    for (colblock=0; colblock<ncols; colblock++) {
        col = colblock*blockDim.y+threadIdx.x;
        if (i<nrows) {
            int cs, tid;
            if (C == blockDim.x) {
                cs = chunkstart[blockIdx.x];
                tid = threadIdx.y;
            } else {
                cs = chunkstart[i/C];
                tid = threadIdx.y%C;
            }
            int j;
            v_t tmp;

            zero<v_t>(tmp);

            for (j=0; j<rowlen[i]; j++) {
                tmp = axpy<v_t,m_t>(tmp, rhs[rhs_lda*mcol[cs + tid + j*C]+col], val[cs+tid+j*C]);
            }

            if (do_shift) {
                tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*i+col],scale2<v_t,float>(shift[0],-1.f));
            }
            if (do_vshift) {
                tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*i+col],scale2<v_t,float>(shift[col],-1.f));
            }
            if (do_scale) {
                tmp = scale<v_t>(alpha,tmp);
            }
            if (do_axpy) {
                lhs[lhs_lda*i+col] = axpy<v_t,float>(lhs[lhs_lda*i+col],tmp,1.f);
            } else if (do_axpby) {
                lhs[lhs_lda*i+col] = axpy<v_t,float>(scale<v_t>(lhs[lhs_lda*i+col],beta),tmp,1.f);
            } else {
                lhs[lhs_lda*i+col] = tmp;
            }
        }
#ifdef LOCALDOT_ONTHEFLY 
        if (do_localdot) {
            v_t dot1, dot2, dot3;
            zero<v_t>(dot1);
            zero<v_t>(dot2);
            zero<v_t>(dot3);

            if (i<nrows) {
                dot1 = axpy<v_t>(dot1,lhs[lhs_lda*i+col],lhs[lhs_lda*i+col]);
                dot2 = axpy<v_t>(dot2,rhs[rhs_lda*i+col],lhs[lhs_lda*i+col]);
                dot3 = axpy<v_t>(dot3,rhs[rhs_lda*i+col],rhs[rhs_lda*i+col]);
            } else {
                zero<v_t>(dot1);
                zero<v_t>(dot2);
                zero<v_t>(dot3);
            }

            dot1 = blockReduceSum(dot1);
            dot2 = blockReduceSum(dot2);
            dot3 = blockReduceSum(dot3);

            if (threadIdx.y==0) {
                localdot[0*ncols*blockDim.y*gridDim.x + col*gridDim.x + blockIdx.x] = dot1;
                localdot[1*ncols*blockDim.y*gridDim.x + col*gridDim.x + blockIdx.x] = dot2;
                localdot[2*ncols*blockDim.y*gridDim.x + col*gridDim.x + blockIdx.x] = dot3;
            }
        }
#endif
    }

}

    template<typename m_t, typename v_t>  
__global__ void SELL_kernel_CU_tmpl(v_t *lhs, int lhs_lda, v_t *rhs, int rhs_lda, ghost_spmv_flags_t flags, int nrows, int nrowspadded, int ncols, ghost_lidx_t *rowlen, ghost_lidx_t *mcol, m_t *val, ghost_lidx_t *chunkstart, ghost_lidx_t *chunklen, int C, int T, v_t *shift, v_t alpha, v_t beta, v_t *localdot, const bool do_axpy, const bool do_axpby, const bool do_scale, const bool do_shift, const bool do_vshift, const bool do_localdot)
{
    UNUSED(T);
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int colblock,col;

    for (colblock=0; colblock<ncols; colblock++) {
        col = colblock*blockDim.y+threadIdx.y;
        if (i<nrows) {
            int cs, tid;
            if (C == blockDim.x) {
                cs = chunkstart[blockIdx.x];
                tid = threadIdx.x;
            } else {
                cs = chunkstart[i/C];
                tid = threadIdx.x%C;
            }
            int j;
            v_t tmp;

            zero<v_t>(tmp);

            for (j=0; j<rowlen[i]; j++) {
                tmp = axpy<v_t,m_t>(tmp, rhs[rhs_lda*col+mcol[cs + tid + j*C]], val[cs+tid+j*C]);
            }

            if (do_shift) {
                tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*col+i],scale2<v_t,float>(shift[0],-1.f));
            }
            if (do_vshift) {
                tmp = axpy<v_t,v_t>(tmp,rhs[rhs_lda*col+i],scale2<v_t,float>(shift[col*blockDim.y+threadIdx.y],-1.f));
            }
            if (do_scale) {
                tmp = scale<v_t>(alpha,tmp);
            }
            if (do_axpy) {
                lhs[lhs_lda*col+i] = axpy<v_t,float>(lhs[lhs_lda*col+i],tmp,1.f);
            } else if (do_axpby) {
                lhs[lhs_lda*col+i] = axpy<v_t,float>(scale<v_t>(lhs[lhs_lda*col+i],beta),tmp,1.f);
            } else {
                lhs[lhs_lda*col+i] = tmp;
            }
        }
#ifdef LOCALDOT_ONTHEFLY 
        if (do_localdot) {
            v_t dot1, dot2, dot3;
            zero<v_t>(dot1);
            zero<v_t>(dot2);
            zero<v_t>(dot3);

            if (i<nrows) {
                dot1 = axpy<v_t>(dot1,lhs[lhs_lda*col+i],lhs[lhs_lda*col+i]);
                dot2 = axpy<v_t>(dot2,rhs[rhs_lda*col+i],lhs[lhs_lda*col+i]);
                dot3 = axpy<v_t>(dot3,rhs[rhs_lda*col+i],rhs[rhs_lda*col+i]);
            } else {
                zero<v_t>(dot1);
                zero<v_t>(dot2);
                zero<v_t>(dot3);
            }

            dot1 = blockReduceSum(dot1);
            dot2 = blockReduceSum(dot2);
            dot3 = blockReduceSum(dot3);

            if (threadIdx.x==0) {
                localdot[0*ncols*blockDim.y*gridDim.x + col*gridDim.x + blockIdx.x] = dot1;
                localdot[1*ncols*blockDim.y*gridDim.x + col*gridDim.x + blockIdx.x] = dot2;
                localdot[2*ncols*blockDim.y*gridDim.x + col*gridDim.x + blockIdx.x] = dot3;
            }
        }
#endif
    }
}

    template<typename m_t, typename v_t>
__global__ void SELLT_kernel_CU_tmpl(v_t *lhs, v_t *rhs, ghost_spmv_flags_t flags, ghost_lidx_t nrows, ghost_lidx_t nrowspadded, ghost_lidx_t *rowlen, ghost_lidx_t *col, m_t *val, ghost_lidx_t *chunkstart, ghost_lidx_t *chunklen, ghost_lidx_t C, int T, v_t *shift, v_t alpha, v_t beta, v_t *localdot, const bool do_axpy, const bool do_axpby, const bool do_scale, const bool do_shift, const bool do_vshift, const bool do_localdot)
{
    int i = threadIdx.x+blockIdx.x*blockDim.x;

    if (i<nrows) {
        int tib = threadIdx.x*blockDim.y+threadIdx.y;
        int cs, tid; // chunk start, thread row in block
        int j;
        v_t tmp;
        v_t *smem = (v_t *)shared;

        if (C == blockDim.x) {
            cs = chunkstart[blockIdx.x];
            tid = threadIdx.x;
        } else {
            cs = chunkstart[i/C];
            tid = threadIdx.x%C;
        }
        zero<v_t>(tmp);


        for (j=0; j<rowlen[i]/T; j++) {
#ifdef SELLT_STRIDE_ONE
            tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + tid + (threadIdx.y*rowlen[i]/T+j)*C]], val[cs + tid + (threadIdx.y*rowlen[i]/T+j)*C]);
#else
            tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + tid + (threadIdx.y+j*blockDim.y)*C]], val[cs + tid + (threadIdx.y+j*blockDim.y)*C]);
#endif
        }

        smem[tib] = tmp;
        __syncthreads();

        if (T>2) {
            if (T>4) {
                if (T>8) {
                    if (T>16) {
                        if (T>32) {
                            if (T>64) {
                                if (threadIdx.y<64) {
                                    smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+64],1.f);
                                    __syncthreads();
                                }
                            }
                            if (threadIdx.y<32) {
                                smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+32],1.f);
                                __syncthreads();
                            }
                        }
                        if (threadIdx.y<16) {
                            smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+16],1.f);
                            __syncthreads();
                        }
                    }
                    if (threadIdx.y<8) {
                        smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+8],1.f);
                        __syncthreads();
                    }
                }
                if (threadIdx.y<4) {
                    smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+4],1.f);
                    __syncthreads();    
                }
            }
            if (threadIdx.y<2) {
                smem[tib] = axpy<v_t,float>(smem[tib],smem[tib+2],1.f);
                __syncthreads();
            }
        }

        if (threadIdx.y == 0) {
            if (do_shift) {
                if (do_scale) {
                    if (do_axpy) {
                        lhs[i] = axpy<v_t,float>(lhs[i],scale<v_t>(alpha,axpy<v_t,v_t>(tmp,rhs[i],scale2<v_t,float>(shift[0],-1.f))),1.f);
                    } else if (do_axpby) {
                        lhs[i] = axpy<v_t,float>(scale<v_t>(lhs[i],beta),scale<v_t>(alpha,axpy<v_t,v_t>(tmp,rhs[i],scale2<v_t,float>(shift[0],-1.f))),1.f);
                    } else {
                        lhs[i] = scale<v_t>(alpha,axpy<v_t,v_t>(tmp,rhs[i],scale2<v_t,float>(shift[0],-1.f)));
                    }
                } else {
                    if (do_axpy) {
                        lhs[i] = axpy<v_t,float>(lhs[i],axpy<v_t,v_t>(tmp,rhs[i],scale2<v_t,float>(shift[0],-1.f)),1.f);
                    } else if (do_axpby) {
                        lhs[i] = axpy<v_t,float>(scale<v_t>(lhs[i],beta),axpy<v_t,v_t>(tmp,rhs[i],scale2<v_t,float>(shift[0],-1.f)),1.f);
                    } else {
                        lhs[i] = axpy<v_t,v_t>(tmp,rhs[i],scale2<v_t,float>(shift[0],-1.f));
                    }
                }
            } else {
                if (do_scale) {
                    if (do_axpy) {
                        lhs[i] = axpy<v_t,float>(lhs[i],scale<v_t>(alpha,tmp),1.f);
                    } else if (do_axpby) {
                        lhs[i] = axpy<v_t,float>(scale<v_t>(lhs[i],beta),scale<v_t>(alpha,tmp),1.f);
                    } else {
                        lhs[i] = scale<v_t>(alpha,tmp);
                    }
                } else {
                    if (do_axpy) {
                        lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
                    } else if (do_axpby) {
                        lhs[i] = axpy<v_t,float>(scale<v_t>(lhs[i],beta),tmp,1.f);
                    } else {
                        lhs[i] = tmp;
                    }
                }

            }
        }
    }
}


extern "C" ghost_error_t dd_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{
    CHOOSE_KERNEL(double,double,double);
}

extern "C" ghost_error_t ds_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(double,float,float);
}

extern "C" ghost_error_t dc_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(double,hipFloatComplex,complex float);
}

extern "C" ghost_error_t dz_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(double,hipDoubleComplex,complex double);
}

extern "C" ghost_error_t sd_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(float,double,double);
}

extern "C" ghost_error_t ss_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(float,float,float);
}

extern "C" ghost_error_t sc_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(float,hipFloatComplex,complex float);
}

extern "C" ghost_error_t sz_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(float,hipDoubleComplex,complex double);
}

extern "C" ghost_error_t zd_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,double,double);
}

extern "C" ghost_error_t zs_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,float,float);
}

extern "C" ghost_error_t zc_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,hipFloatComplex,complex float);
}

extern "C" ghost_error_t zz_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipDoubleComplex,hipDoubleComplex,complex double);
}

extern "C" ghost_error_t cd_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipFloatComplex,double,double);
}

extern "C" ghost_error_t cs_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipFloatComplex,float,float);
}

extern "C" ghost_error_t cc_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipFloatComplex,hipFloatComplex,complex float);
}

extern "C" ghost_error_t cz_SELL_kernel_CU(ghost_sparsemat_t *mat, ghost_densemat_t *lhs, ghost_densemat_t *rhs, ghost_spmv_flags_t flags, va_list argp)
{ 
    CHOOSE_KERNEL(hipFloatComplex,hipDoubleComplex,complex double);
}

