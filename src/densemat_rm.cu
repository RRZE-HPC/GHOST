#include "hip/hip_runtime.h"
#include "ghost/config.h"
#include "ghost/types.h"
#include "ghost/util.h"
#include "ghost/densemat_rm.h"
#include "ghost/log.h"
#include "ghost/timing.h"
#include "ghost/locality.h"
#include "ghost/instr.h"
#include "ghost/rand.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>
#include <sys/types.h>
#include <unistd.h>
#include <complex.h>

#include "ghost/cu_complex.h"


#define ROWMAJOR
#include "ghost/densemat_common.cu.def"

