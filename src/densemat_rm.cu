#include "hip/hip_runtime.h"
#include "ghost/config.h"
#include "ghost/types.h"
#include "ghost/util.h"
#include "ghost/densemat_rm.h"
#include "ghost/log.h"
#include "ghost/timing.h"
#include "ghost/locality.h"
#include "ghost/instr.h"
#include "ghost/rand.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>
#include <sys/types.h>
#include <unistd.h>
#include <complex.h>

#include "ghost/cu_complex.h"


#define THREADSPERBLOCK 1024


template<typename T>  
__global__ static void cu_vaxpby_kernel(T *v1, T *v2, T *a, T *b, ghost_lidx_t nrows, ghost_lidx_t ncols, ghost_lidx_t ld1, ghost_lidx_t ld2)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        ghost_lidx_t v;
        for (v=0; v<ncols; v++) {
            v1[idx*ld1+v] = axpby<T>(v2[idx*ld2+v],v1[idx*ld1+v],a[v],b[v]);
        }
    }
}

template<typename T>  
__global__ static void cu_axpby_kernel(T *v1, T *v2, T a, T b, ghost_lidx_t nrows, ghost_lidx_t ncols, ghost_lidx_t ld1, ghost_lidx_t ld2) 
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        ghost_lidx_t v;
        for (v=0; v<ncols; v++) {
            v1[idx*ld1+v] = axpby<T>(v2[idx*ld2+v],v1[idx*ld1+v],a,b);
        }
    }
}

template<typename T>  
__global__ static void cu_scale_kernel(T *vec, T a, ghost_lidx_t nrows, ghost_lidx_t ncols, ghost_lidx_t ld)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        ghost_lidx_t v;
        for (v=0; v<ncols; v++) {
            vec[idx*ld+v] = scale<T>(a,vec[idx*ld+v]);
        }
    }

}

template<typename T>  
__global__ static void cu_vscale_kernel(T *vec, T *a, ghost_lidx_t nrows, ghost_lidx_t ncols, ghost_lidx_t ld)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        ghost_lidx_t v;
        for (v=0; v<ncols; v++) {
            vec[idx*ld+v] = scale<T>(a[v],vec[idx*ld+v]);
        }
    }
}

template<typename T>  
__global__ static void cu_fromscalar_kernel(T *vec, T a, ghost_lidx_t nrows, ghost_lidx_t ncols, ghost_lidx_t ld)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x) {
        ghost_lidx_t v;
        for (v=0; v<ncols; v++) {
            vec[idx*ld+v] = a;
        }
    }
}


template<typename T>  
__global__ static void cu_communicationassembly_kernel(T *vec, T *work, ghost_lidx_t offs, ghost_lidx_t *duelist, ghost_lidx_t ncols, ghost_lidx_t ndues, ghost_lidx_t ncolspadded, ghost_lidx_t *perm)
{
    int due = blockIdx.x*blockDim.x+threadIdx.x;
    int col = threadIdx.y;

    if (perm) {
        for (;due < ndues; due+=gridDim.x*blockDim.x) {
            work[(offs+due)*ncols+col] = vec[perm[duelist[due]]*ncolspadded+col];
        }
    } else {
        for (;due < ndues; due+=gridDim.x*blockDim.x) {
            work[(offs+due)*ncols+col] = vec[duelist[due]*ncolspadded+col];
        }
    }
}

extern "C" ghost_error_t ghost_densemat_rm_cu_communicationassembly(void * work, ghost_lidx_t *dueptr, ghost_densemat_t *vec, ghost_lidx_t *perm)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
  
    if (!vec->context->cu_duelist) {
       ERROR_LOG("cu_duelist must not be NULL!");
       return GHOST_ERR_INVALID_ARG;
    }
    if (!dueptr) {
       ERROR_LOG("dueptr must not be NULL!");
       return GHOST_ERR_INVALID_ARG;
    }


    int nrank, proc, me;
    ghost_context_t *ctx = vec->context;
    
    ghost_nrank(&nrank,ctx->mpicomm); 
    ghost_rank(&me,ctx->mpicomm);
            
    for (proc=0 ; proc<nrank ; proc++){
        dim3 block((int)ceil((double)THREADSPERBLOCK/vec->traits.ncols),vec->traits.ncols);
        dim3 grid((int)ceil((double)ctx->dues[proc]/block.x));
        DEBUG_LOG(1,"communication assembly with grid %d block %dx%d %d->%d",grid.x,block.x,block.y,me,proc);
        if (vec->traits.datatype & GHOST_DT_COMPLEX)
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                if (ctx->dues[proc]) {
                    cu_communicationassembly_kernel<hipDoubleComplex><<< grid,block >>>((hipDoubleComplex *)vec->cu_val, ((hipDoubleComplex *)work),dueptr[proc],ctx->cu_duelist[proc],vec->traits.ncols,ctx->dues[proc],vec->stride,perm);
                }
            } 
            else 
            {
                if (ctx->dues[proc]) {
                    cu_communicationassembly_kernel<hipFloatComplex><<< grid,block >>>((hipFloatComplex *)vec->cu_val, ((hipFloatComplex *)work),dueptr[proc],ctx->cu_duelist[proc],vec->traits.ncols,ctx->dues[proc],vec->stride,perm);
                }
            }
        }
        else
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                if (ctx->dues[proc]) {
                    cu_communicationassembly_kernel<double><<< grid,block >>>((double *)vec->cu_val, ((double *)work),dueptr[proc],ctx->cu_duelist[proc],vec->traits.ncols,ctx->dues[proc],vec->stride,perm);
                }
            } 
            else 
            {
                if (ctx->dues[proc]) {
                    cu_communicationassembly_kernel<float><<< grid,block >>>((float *)vec->cu_val, ((float *)work),dueptr[proc],ctx->cu_duelist[proc],vec->traits.ncols,ctx->dues[proc],vec->stride,perm);
                }
            }
        }
    }
    hipDeviceSynchronize();

    if (hipPeekAtLastError() != hipSuccess) {
        ERROR_LOG("Error in kernel");
        return GHOST_ERR_CUDA;
    }
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return GHOST_SUCCESS;

}

extern "C" ghost_error_t ghost_densemat_rm_cu_vaxpy(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS; 
    
    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            complex double *one;
            GHOST_CALL_RETURN(ghost_malloc((void **)&one,v1->traits.ncols*sizeof(complex double)));
            int v;
            for (v=0; v<v1->traits.ncols; v++) {
                one[v] = 1.+I*0.;
            }
            ret =  ghost_densemat_rm_cu_vaxpby(v1,v2,a,one);
        } 
        else 
        {
            complex float *one;
            GHOST_CALL_RETURN(ghost_malloc((void **)&one,v1->traits.ncols*sizeof(complex float)));
            int v;
            for (v=0; v<v1->traits.ncols; v++) {
                one[v] = 1.+I*0.;
            }
            ret =  ghost_densemat_rm_cu_vaxpby(v1,v2,a,one);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            double *one;
            GHOST_CALL_RETURN(ghost_malloc((void **)&one,v1->traits.ncols*sizeof(double)));
            int v;
            for (v=0; v<v1->traits.ncols; v++) {
                one[v] = 1.;
            }
            ret =  ghost_densemat_rm_cu_vaxpby(v1,v2,a,one);
        } 
        else 
        {
            float *one;
            GHOST_CALL_RETURN(ghost_malloc((void **)&one,v1->traits.ncols*sizeof(float)));
            int v;
            for (v=0; v<v1->traits.ncols; v++) {
                one[v] = 1.;
            }
            ret =  ghost_densemat_rm_cu_vaxpby(v1,v2,a,one);
        }
    }
    
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}
    
extern "C" ghost_error_t ghost_densemat_rm_cu_vaxpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot VAXPBY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;

    void *d_a;
    void *d_b;
    size_t sizeofdt;

    ghost_datatype_size(&sizeofdt,v1->traits.datatype);
    
    GHOST_CALL_GOTO(ghost_cu_malloc(&d_a,v1->traits.ncols*sizeofdt),err,ret);
    GHOST_CALL_GOTO(ghost_cu_malloc(&d_b,v1->traits.ncols*sizeofdt),err,ret);
   
    ghost_lidx_t c; 
    for (c=0; c<v1->traits.ncols; c++) {
            GHOST_CALL_GOTO(ghost_cu_upload(&((char *)d_a)[c*sizeofdt],&((char *)a)[c*sizeofdt],sizeofdt),err,ret);
            GHOST_CALL_GOTO(ghost_cu_upload(&((char *)d_b)[c*sizeofdt],&((char *)b)[c*sizeofdt],sizeofdt),err,ret);
    }
    
    
    void *v1val, *v2val;
    ghost_densemat_t *v1compact, *v2compact;
    
    if (v1->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) v1 before operation");
        GHOST_CALL_GOTO(v1->clone(v1,&v1compact,v1->traits.nrows,0,v1->traits.ncols,0),err,ret);
    } else {
        v1compact = v1;
    }
    if (v2->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) v2 before operation");
        GHOST_CALL_GOTO(v2->clone(v2,&v2compact,v2->traits.nrows,0,v2->traits.ncols,0),err,ret);
    } else {
        v2compact = v2;
    }
    GHOST_CALL_GOTO(ghost_densemat_cu_valptr(v1compact,&v1val),err,ret);
    GHOST_CALL_GOTO(ghost_densemat_cu_valptr(v2compact,&v2val),err,ret);
    

    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipDoubleComplex *)v1val, (hipDoubleComplex *)v2val,(hipDoubleComplex *)d_a,(hipDoubleComplex *)d_b,
                 v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
        } 
        else 
        {
            cu_vaxpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipFloatComplex *)v1val, (hipFloatComplex *)v2val,(hipFloatComplex *)d_a,(hipFloatComplex *)d_b,
                 v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (double *)v1val, (double *)v2val,(double *)d_a,(double *)d_b,
                 v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
        } 
        else 
        {
            cu_vaxpby_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (float *)v1val, (float *)v2val,(float *)d_a,(float *)d_b,
                 v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
        }
    }
    if (v1compact != v1) {
        GHOST_CALL_GOTO(v1->fromVec(v1,v1compact,0,0),err,ret);
        v1compact->destroy(v1compact);
    }
    if (v2compact != v2) {
        v2compact->destroy(v2compact);
    }
    
    goto out;
err:
out:
    GHOST_CALL_RETURN(ghost_cu_free(d_a));
    GHOST_CALL_RETURN(ghost_cu_free(d_b));
    hipDeviceSynchronize();
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_dotprod(ghost_densemat_t *vec, void *res, ghost_densemat_t *vec2)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;
    
    if (vec->traits.datatype != vec2->traits.datatype)
    {
        ERROR_LOG("Cannot DOT vectors with different data types (%s and %s)",ghost_datatype_string(vec->traits.datatype),ghost_datatype_string(vec2->traits.datatype));
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,vec->traits.datatype);
    ghost_densemat_t *veccompact;
    ghost_densemat_t *vec2compact;

    if (vec->traits.flags & GHOST_DENSEMAT_VIEW) {
        INFO_LOG("Cloning (and compressing) vec1 before dotproduct");
        vec->clone(vec,&veccompact,vec->traits.nrows,0,vec->traits.ncols,0);
    } else {
        veccompact = vec;
    }
    if (vec2->traits.flags & GHOST_DENSEMAT_VIEW) {
        INFO_LOG("Cloning (and compressing) vec2 before dotproduct");
        vec2->clone(vec2,&vec2compact,vec2->traits.nrows,0,vec2->traits.ncols,0);
    } else {
        vec2compact = vec2;
    }
  
     
    hipblasHandle_t ghost_cublas_handle;
    GHOST_CALL_GOTO(ghost_cu_cublas_handle(&ghost_cublas_handle),err,ret); 
    ghost_lidx_t v;
    for (v=0; v<veccompact->traits.ncols; v++)
    {
        char *v1 = veccompact->cu_val+v*veccompact->elSize;
        char *v2 = vec2compact->cu_val+v*veccompact->elSize;
        if (vec->traits.datatype & GHOST_DT_COMPLEX)
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_GOTO(hipblasZdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipDoubleComplex *)v1,veccompact->stride,(const hipDoubleComplex *)v2,vec2compact->stride,&((hipDoubleComplex *)res)[v]),err,ret);
            } 
            else 
            {
                CUBLAS_CALL_GOTO(hipblasCdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipFloatComplex *)v1,veccompact->stride,(const hipFloatComplex *)v2,vec2compact->stride,&((hipFloatComplex *)res)[v]),err,ret);
            }
        }
        else
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_GOTO(hipblasDdot(ghost_cublas_handle,vec->traits.nrows,
                            (const double *)v1,veccompact->stride,(const double *)v2,vec2compact->stride,&((double *)res)[v]),err,ret);
            } 
            else 
            {
                CUBLAS_CALL_GOTO(hipblasSdot(ghost_cublas_handle,vec->traits.nrows,
                            (const float *)v1,veccompact->stride,(const float *)v2,vec2compact->stride,&((float *)res)[v]),err,ret);
            }
        }
    }
    if (veccompact != vec) {
        veccompact->destroy(veccompact);
    }
    if (vec2compact != vec2) {
        vec2compact->destroy(vec2compact);
    }

    goto out;
err:
out:
    hipDeviceSynchronize();
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_axpy(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS; 
    
    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            const hipDoubleComplex one = make_hipDoubleComplex(1.,0);
            ret =  ghost_densemat_rm_cu_axpby(v1,v2,a,(void *)&one);
        } 
        else 
        {
            const hipFloatComplex one = make_hipFloatComplex(1.,0.);
            ret = ghost_densemat_rm_cu_axpby(v1,v2,a,(void *)&one);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            const double one = 1.;
            ret = ghost_densemat_rm_cu_axpby(v1,v2,a,(void *)&one);
        } 
        else 
        {
            const float one = 1.f;
            ret = ghost_densemat_rm_cu_axpby(v1,v2,a,(void *)&one);
        }
    }
    
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_axpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot AXPBY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;
    void *v1val, *v2val;
    ghost_densemat_t *v1compact, *v2compact;
    
    if (v1->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) v1 before operation");
        GHOST_CALL_GOTO(v1->clone(v1,&v1compact,v1->traits.nrows,0,v1->traits.ncols,0),err,ret);
    } else {
        v1compact = v1;
    }
    if (v2->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) v2 before operation");
        GHOST_CALL_GOTO(v2->clone(v2,&v2compact,v2->traits.nrows,0,v2->traits.ncols,0),err,ret);
    } else {
        v2compact = v2;
    }
    GHOST_CALL_GOTO(ghost_densemat_cu_valptr(v1compact,&v1val),err,ret);
    GHOST_CALL_GOTO(ghost_densemat_cu_valptr(v2compact,&v2val),err,ret);

    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipDoubleComplex *)v1val, (hipDoubleComplex *)v2val,*((hipDoubleComplex *)a),*((hipDoubleComplex *)b),v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
        } 
        else 
        {
            cu_axpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipFloatComplex *)v1val, (hipFloatComplex *)v2val,*((hipFloatComplex *)a),*((hipFloatComplex *)b),v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
            
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<double><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((double *)v1val, (double *)v2val,*((double *)a),*((double *)b),v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
        } 
        else 
        {
            cu_axpby_kernel<float><<< (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((float *)v1val, (float *)v2val,*((float *)a),*((float *)b),v1->traits.nrows,v1->traits.ncols,v1->stride,v2->stride);
        }
    }
    if (v1compact != v1) {
        GHOST_CALL_GOTO(v1->fromVec(v1,v1compact,0,0),err,ret);
        v1compact->destroy(v1compact);
    }
    if (v2compact != v2) {
        v2compact->destroy(v2compact);
    }

    goto out;
err:
out:
    hipDeviceSynchronize();
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_scale(ghost_densemat_t *vec, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;
    
    void *vecval;
    ghost_densemat_t *veccompact;
    
    if (vec->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) vec before operation");
        GHOST_CALL_GOTO(vec->clone(vec,&veccompact,vec->traits.nrows,0,vec->traits.ncols,0),err,ret);
    } else {
        veccompact = vec;
    }
    GHOST_CALL_GOTO(ghost_densemat_cu_valptr(veccompact,&vecval),err,ret);
    
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_scale_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vecval, *(hipDoubleComplex *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        } 
        else 
        {
            cu_scale_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vecval, *(hipFloatComplex *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_scale_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vecval, *(double *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        } 
        else 
        {
            cu_scale_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vecval, *(float *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        }
    }
    if (veccompact != vec) {
        INFO_LOG("Transform back");
        GHOST_CALL_GOTO(vec->fromVec(vec,veccompact,0,0),err,ret);
        veccompact->destroy(veccompact);
    }
    
    goto out;

err:

out:
    hipDeviceSynchronize();
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    
    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_vscale(ghost_densemat_t *vec, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;

    void *d_a;
    ghost_idx_t c;
    void *vecval;
    ghost_densemat_t *veccompact;
    
    if (vec->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) vec before operation");
        GHOST_CALL_GOTO(vec->clone(vec,&veccompact,vec->traits.nrows,0,vec->traits.ncols,0),err,ret);
    } else {
        veccompact = vec;
    }
    GHOST_CALL_GOTO(ghost_densemat_cu_valptr(veccompact,&vecval),err,ret);

    GHOST_CALL_GOTO(ghost_cu_malloc(&d_a,vec->traits.ncols*vec->elSize),err,ret);
    
    for (c=0; c<vec->traits.ncols; c++) {
        GHOST_CALL_GOTO(ghost_cu_upload(&((char *)d_a)[c*vec->elSize],&((char *)a)[c*vec->elSize],vec->elSize),err,ret);
    }
    
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vecval, (hipDoubleComplex *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        } 
        else 
        {
            cu_vscale_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vecval, (hipFloatComplex *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vecval, (double *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        } 
        else 
        {
            cu_vscale_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vecval, (float *)d_a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        }
    }
    if (veccompact != vec) {
        INFO_LOG("Transform back");
        GHOST_CALL_GOTO(vec->fromVec(vec,veccompact,0,0),err,ret);
        veccompact->destroy(veccompact);
    }

    goto out;
err:
out:
    hipDeviceSynchronize();
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_fromScalar(ghost_densemat_t *vec, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_INITIALIZATION);
    ghost_error_t ret = GHOST_SUCCESS;
    int needInit = 0;
    ghost_densemat_rm_malloc(vec,&needInit);
    
    void *vecval;
    ghost_densemat_t *veccompact;
    
    if (vec->traits.flags & GHOST_DENSEMAT_SCATTERED) {
        INFO_LOG("Cloning (and compressing) vec before operation");
        GHOST_CALL_GOTO(vec->clone(vec,&veccompact,vec->traits.nrows,0,vec->traits.ncols,0),err,ret);
    } else {
        veccompact = vec;
    }
    GHOST_CALL_GOTO(ghost_densemat_cu_valptr(veccompact,&vecval),err,ret);

    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vecval, *(hipDoubleComplex *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        } 
        else 
        {
            cu_fromscalar_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vecval, *(hipFloatComplex *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<double><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vecval, *(double *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        } 
        else 
        {
            cu_fromscalar_kernel<float><<< (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vecval, *(float *)a,
                    vec->traits.nrows,vec->traits.ncols,vec->stride);
        }
    }
    if (veccompact != vec) {
        INFO_LOG("Transform back");
        GHOST_CALL_GOTO(vec->fromVec(vec,veccompact,0,0),err,ret);
        veccompact->destroy(veccompact);
    }
    
    goto out;
err:
out:
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_INITIALIZATION);
    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_fromRand(ghost_densemat_t *vec)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_INITIALIZATION);
    ghost_error_t ret = GHOST_SUCCESS;

    ghost_densemat_t *onevec;
    long pid = getpid();
    double time;
    double one[] = {1.,1.};
    float fone[] = {1.,0.};
    double minusahalf[] = {-0.5,0.};
    float fminusahalf[] = {-0.5,0.};
    
    ghost_timing_wcmilli(&time);
    int needInit = 0;
    ghost_densemat_rm_malloc(vec,&needInit);
    hiprandGenerator_t gen;
    CURAND_CALL_GOTO(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT),err,ret);
    CURAND_CALL_GOTO(hiprandSetPseudoRandomGeneratorSeed(gen,ghost_rand_cu_seed_get()),err,ret);

    vec->clone(vec,&onevec,vec->traits.nrows,0,vec->traits.ncols,0);
    onevec->fromScalar(onevec,one);

    one[1] = 0.;
    void *valptr;
    ghost_densemat_t *compactvec;

    if ((vec->traits.ncolsorig != vec->traits.ncols) || (vec->traits.flags & GHOST_DENSEMAT_SCATTERED)) {
        INFO_LOG("Cloning (and compressing) vec before operation");
        vec->clone(vec,&compactvec,vec->traits.nrows,0,vec->traits.ncols,0);
    } else {
        compactvec = vec;
    }
    ghost_densemat_cu_valptr(compactvec,&valptr);


    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CURAND_CALL_GOTO(hiprandGenerateUniformDouble(gen,
                        (double *)valptr,
                        compactvec->traits.nrowspadded*compactvec->traits.ncols*2),err,ret);
        } 
        else 
        {
            CURAND_CALL_GOTO(hiprandGenerateUniform(gen,
                        (float *)valptr,
                        compactvec->traits.nrowspadded*compactvec->traits.ncols*2),err,ret);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CURAND_CALL_GOTO(hiprandGenerateUniformDouble(gen,
                        (double *)valptr,
                        compactvec->traits.nrowspadded*compactvec->traits.ncols),err,ret);
        } 
        else 
        {
            CURAND_CALL_GOTO(hiprandGenerateUniform(gen,
                        (float *)valptr,
                        compactvec->traits.nrowspadded*compactvec->traits.ncols),err,ret);
        }
    }
    if (compactvec->traits.datatype & GHOST_DT_DOUBLE) {
        compactvec->axpby(compactvec,onevec,minusahalf,one);
    } else {
        compactvec->axpby(compactvec,onevec,fminusahalf,fone);
    }
    if (compactvec != vec) {
        vec->fromVec(vec,compactvec,0,0);
        compactvec->destroy(compactvec);
    }
    goto out;
err:
out:
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_INITIALIZATION);
    CURAND_CALL_RETURN(hiprandDestroyGenerator(gen));
    onevec->destroy(onevec);

    return ret;
}
#if 0

extern "C" ghost_error_t ghost_densemat_rm_cu_vaxpy(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;
    void *d_a;
    size_t sizeofdt;
    char colfield[v1->traits.ncolsorig];
    char rowfield[v1->traits.nrowsorig];
    char *cucolfield = NULL, *curowfield = NULL;
    int grid = (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK);
    dim3 block (THREADSPERBLOCK/v1->traits.ncolsorig,v1->traits.ncolsorig);
    ghost_datatype_size(&sizeofdt,v1->traits.datatype);
    
    GHOST_CALL_GOTO(ghost_cu_malloc(&d_a,v1->traits.ncols*sizeofdt),err,ret);
    
    ghost_cu_upload(d_a,a,v1->traits.ncols*sizeofdt);
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot VAXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }

    
    if (ghost_bitmap_weight(v1->ldmask) != v1->traits.ncolsorig || 
            ghost_bitmap_weight(v1->trmask) != v1->traits.nrowsorig ||
            ghost_bitmap_weight(v2->ldmask) != v2->traits.ncolsorig ||
            ghost_bitmap_weight(v2->trmask) != v2->traits.nrowsorig) { 
        
        if (!ghost_bitmap_isequal(v1->ldmask,v2->ldmask) || !ghost_bitmap_isequal(v1->trmask,v2->trmask)) {
            ERROR_LOG("The masks have to be equal!");
            ret = GHOST_ERR_INVALID_ARG;
            goto err;
        }
       
        WARNING_LOG("Potentially slow VAXPY operation because some rows or columns are masked out!");
        
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&cucolfield,v1->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&curowfield,v1->traits.nrowsorig),err,ret);

        ghost_densemat_mask2charfield(v1->ldmask,v1->traits.ncolsorig,colfield);
        ghost_densemat_mask2charfield(v1->trmask,v1->traits.nrowsorig,rowfield);

        GHOST_CALL_GOTO(ghost_cu_upload(cucolfield,colfield,v1->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_upload(curowfield,rowfield,v1->traits.nrowsorig),err,ret);
    }

    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpy_kernel<hipDoubleComplex><<< grid,block >>>((hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,(hipDoubleComplex *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        } 
        else 
        {
            cu_vaxpy_kernel<hipFloatComplex><<< grid,block >>>((hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,(hipFloatComplex *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpy_kernel<double><<< grid,block >>>((double *)v1->cu_val, (double *)v2->cu_val,(double *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        } 
        else 
        {
            cu_vaxpy_kernel<float><<< grid,block >>>((float *)v1->cu_val, (float *)v2->cu_val,(float *)d_a,v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        }
    }
    
    goto out;
err:
out:
    GHOST_CALL_RETURN(ghost_cu_free(cucolfield));
    GHOST_CALL_RETURN(ghost_cu_free(curowfield));
    GHOST_CALL_RETURN(ghost_cu_free(d_a));
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}
    
extern "C" ghost_error_t ghost_densemat_rm_cu_vaxpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;

    void *d_a;
    void *d_b;
    size_t sizeofdt;
    char colfield[v1->traits.ncolsorig];
    char rowfield[v1->traits.nrowsorig];

    char *cucolfield = NULL, *curowfield = NULL;
    int grid = (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK);
    dim3 block (THREADSPERBLOCK/v1->traits.ncolsorig,v1->traits.ncolsorig);
    
    ghost_datatype_size(&sizeofdt,v1->traits.datatype);
    
    GHOST_CALL_GOTO(ghost_cu_malloc(&d_a,v1->traits.ncols*sizeofdt),err,ret);
    GHOST_CALL_GOTO(ghost_cu_malloc(&d_b,v1->traits.ncols*sizeofdt),err,ret);
    
    ghost_cu_upload(d_b,b,v1->traits.ncols*sizeofdt);
    
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot VAXPBY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    
    if (ghost_bitmap_weight(v1->ldmask) != v1->traits.ncolsorig || 
            ghost_bitmap_weight(v1->trmask) != v1->traits.nrowsorig ||
            ghost_bitmap_weight(v2->ldmask) != v2->traits.ncolsorig ||
            ghost_bitmap_weight(v2->trmask) != v2->traits.nrowsorig) { 
        
        if (!ghost_bitmap_isequal(v1->ldmask,v2->ldmask) || !ghost_bitmap_isequal(v1->trmask,v2->trmask)) {
            ERROR_LOG("The masks have to be equal!");
            ret = GHOST_ERR_INVALID_ARG;
            goto err;
        }
       
        WARNING_LOG("Potentially slow VAXPBY operation because some rows or columns are masked out!");
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&cucolfield,v1->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&curowfield,v1->traits.nrowsorig),err,ret);

        ghost_densemat_mask2charfield(v1->ldmask,v1->traits.ncolsorig,colfield);
        ghost_densemat_mask2charfield(v1->trmask,v1->traits.nrowsorig,rowfield);

        GHOST_CALL_GOTO(ghost_cu_upload(cucolfield,colfield,v1->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_upload(curowfield,rowfield,v1->traits.nrowsorig),err,ret);
    }

    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<hipDoubleComplex><<< grid,block >>>(
                (hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,(hipDoubleComplex *)d_a,(hipDoubleComplex *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        } 
        else 
        {
            cu_vaxpby_kernel<hipFloatComplex><<< grid,block >>>(
                (hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,(hipFloatComplex *)d_a,(hipFloatComplex *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vaxpby_kernel<double><<< grid,block >>>(
                 (double *)v1->cu_val, (double *)v2->cu_val,(double *)d_a,(double *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        } 
        else 
        {
            cu_vaxpby_kernel<float><<< grid,block >>>(
                (float *)v1->cu_val, (float *)v2->cu_val,(float *)d_a,(float *)d_b,
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        }
    }
    goto out;
err:
out:
    GHOST_CALL_RETURN(ghost_cu_free(cucolfield));
    GHOST_CALL_RETURN(ghost_cu_free(curowfield));
    GHOST_CALL_RETURN(ghost_cu_free(d_a));
    GHOST_CALL_RETURN(ghost_cu_free(d_a));
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_dotprod(ghost_densemat_t *vec, void *res, ghost_densemat_t *vec2)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;
    
    if (vec->traits.datatype != vec2->traits.datatype)
    {
        ERROR_LOG("Cannot DOT vectors with different data types (%s and %s)",ghost_datatype_string(vec->traits.datatype),ghost_datatype_string(vec2->traits.datatype));
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    size_t sizeofdt;
    ghost_datatype_size(&sizeofdt,vec->traits.datatype);
    ghost_densemat_t *vecclone;
    ghost_densemat_t *vec2clone;

    if (ghost_bitmap_weight(vec->ldmask) != vec->traits.ncolsorig || 
            ghost_bitmap_weight(vec->trmask) != vec->traits.nrowsorig) {
        INFO_LOG("Cloning (and compressing) vec1 before dotproduct");
        vec->clone(vec,&vecclone,vec->traits.nrows,0,vec->traits.ncols,0);
    } else {
        vecclone = vec;
    }
    if (ghost_bitmap_weight(vec2->ldmask) != vec2->traits.ncolsorig || 
            ghost_bitmap_weight(vec2->trmask) != vec2->traits.nrowsorig) {
        INFO_LOG("Cloning (and compressing) vec1 before dotproduct");
        vec2->clone(vec2,&vec2clone,vec2->traits.nrows,0,vec2->traits.ncols,0);
    } else {
        vec2clone = vec2;
    }
  
     
    hipblasHandle_t ghost_cublas_handle;
    GHOST_CALL_GOTO(ghost_cu_cublas_handle(&ghost_cublas_handle),err,ret); 
    ghost_lidx_t v;
    for (v=0; v<vecclone->traits.ncols; v++)
    {
        char *v1 = &((char *)(vecclone->cu_val))[v*sizeofdt];
        char *v2 = &((char *)(vec2clone->cu_val))[v*sizeofdt];
        if (vec->traits.datatype & GHOST_DT_COMPLEX)
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_GOTO(hipblasZdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipDoubleComplex *)v1,vec->stride,(const hipDoubleComplex *)v2,vec2->stride,&((hipDoubleComplex *)res)[v]),err,ret);
            } 
            else 
            {
                CUBLAS_CALL_GOTO(hipblasCdotc(ghost_cublas_handle,vec->traits.nrows,
                            (const hipFloatComplex *)v1,vec->stride,(const hipFloatComplex *)v2,vec2->stride,&((hipFloatComplex *)res)[v]),err,ret);
            }
        }
        else
        {
            if (vec->traits.datatype & GHOST_DT_DOUBLE)
            {
                CUBLAS_CALL_GOTO(hipblasDdot(ghost_cublas_handle,vec->traits.nrows,
                            (const double *)v1,vec->stride,(const double *)v2,vec2->stride,&((double *)res)[v]),err,ret);
            } 
            else 
            {
                CUBLAS_CALL_GOTO(hipblasSdot(ghost_cublas_handle,vec->traits.nrows,
                            (const float *)v1,vec->stride,(const float *)v2,vec2->stride,&((float *)res)[v]),err,ret);
            }
        }
    }

    goto out;
err:
out:
    if (!ghost_bitmap_iscompact(vec->ldmask) || 
            !ghost_bitmap_iscompact(vec->trmask)) {
        vecclone->destroy(vecclone);
    }
    
    if (!ghost_bitmap_iscompact(vec2->ldmask) || 
            !ghost_bitmap_iscompact(vec2->trmask)) {
        vec2clone->destroy(vec2clone);
    }
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_axpy(ghost_densemat_t *vec, ghost_densemat_t *vec2, void *a)
{
    if (vec->traits.datatype != vec2->traits.datatype)
    {
        ERROR_LOG("Cannot AXPY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;
    
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];

    char *cucolfield = NULL, *curowfield = NULL;
    int grid = (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK);
    dim3 block (THREADSPERBLOCK/vec->traits.ncolsorig,vec->traits.ncolsorig); 

    if (ghost_bitmap_weight(vec->ldmask) != vec->traits.ncolsorig || 
            ghost_bitmap_weight(vec->trmask) != vec->traits.nrowsorig ||
            ghost_bitmap_weight(vec2->ldmask) != vec2->traits.ncolsorig ||
            ghost_bitmap_weight(vec2->trmask) != vec2->traits.nrowsorig) {

        if (!ghost_bitmap_isequal(vec->ldmask,vec2->ldmask) || !ghost_bitmap_isequal(vec->trmask,vec2->trmask)) {
            ERROR_LOG("The masks have to be equal!");
            ret = GHOST_ERR_INVALID_ARG;
            goto err;
        }
       
        WARNING_LOG("Potentially slow AXPY operation because some rows or columns are masked out!");
        
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig),err,ret);

        ghost_densemat_mask2charfield(vec->ldmask,vec->traits.ncolsorig,colfield);
        ghost_densemat_mask2charfield(vec->trmask,vec->traits.nrowsorig,rowfield);

        GHOST_CALL_GOTO(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig),err,ret);
    }

    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            const hipDoubleComplex one = make_hipDoubleComplex(1.,1.);
            cu_axpby_kernel<hipDoubleComplex><<< grid,block >>>
                ((hipDoubleComplex *)vec->cu_val, (hipDoubleComplex *)vec2->cu_val,*((hipDoubleComplex *)a),one,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            const hipFloatComplex one = make_hipFloatComplex(1.,1.);
            cu_axpby_kernel<hipFloatComplex><<< grid,block >>>
                ((hipFloatComplex *)vec->cu_val, (hipFloatComplex *)vec2->cu_val,*((hipFloatComplex *)a),one,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<double><<< grid,block >>>
                ((double *)vec->cu_val, (double *)vec2->cu_val,*((double *)a),(double)1.,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            cu_axpby_kernel<float><<< grid,block >>>
                ((float *)vec->cu_val, (float *)vec2->cu_val,*((float *)a),(float)1.,
                 vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }

    
    goto out;
err:
out:
    
    GHOST_CALL_RETURN(ghost_cu_free(cucolfield));
    GHOST_CALL_RETURN(ghost_cu_free(curowfield));
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_axpby(ghost_densemat_t *v1, ghost_densemat_t *v2, void *a, void *b)
{
    if (v1->traits.datatype != v2->traits.datatype)
    {
        ERROR_LOG("Cannot AXPBY vectors with different data types");
        return GHOST_ERR_NOT_IMPLEMENTED;
    }
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;

    char colfield[v1->traits.ncolsorig];
    char rowfield[v1->traits.nrowsorig];

    char *cucolfield = NULL, *curowfield = NULL;
    int grid = (int)ceil((double)v1->traits.nrows/THREADSPERBLOCK);
    dim3 block (THREADSPERBLOCK/v1->traits.ncolsorig,v1->traits.ncolsorig); 
    INFO_LOG("block %dx%d",block.x,block.y);

    if (ghost_bitmap_weight(v1->ldmask) != v1->traits.ncolsorig || 
            ghost_bitmap_weight(v1->trmask) != v1->traits.nrowsorig ||
            ghost_bitmap_weight(v2->ldmask) != v2->traits.ncolsorig ||
            ghost_bitmap_weight(v2->trmask) != v2->traits.nrowsorig) { 
        
        if (!ghost_bitmap_isequal(v1->ldmask,v2->ldmask) || !ghost_bitmap_isequal(v1->trmask,v2->trmask)) {
            ERROR_LOG("The masks have to be equal!");
            ret = GHOST_ERR_INVALID_ARG;
            goto err;
        }
        
        WARNING_LOG("Potentially slow AXPBY operation because some rows or columns are masked out!");

        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&cucolfield,v1->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&curowfield,v1->traits.nrowsorig),err,ret);

        ghost_densemat_mask2charfield(v1->ldmask,v1->traits.ncolsorig,colfield);
        ghost_densemat_mask2charfield(v1->trmask,v1->traits.nrowsorig,rowfield);

        GHOST_CALL_GOTO(ghost_cu_upload(cucolfield,colfield,v1->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_upload(curowfield,rowfield,v1->traits.nrowsorig),err,ret);
    }


    if (v1->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<hipDoubleComplex><<< grid,block >>>
                ((hipDoubleComplex *)v1->cu_val, (hipDoubleComplex *)v2->cu_val,*((hipDoubleComplex *)a),*((hipDoubleComplex *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        } 
        else 
        {
            cu_axpby_kernel<hipFloatComplex><<< grid,block >>>
                ((hipFloatComplex *)v1->cu_val, (hipFloatComplex *)v2->cu_val,*((hipFloatComplex *)a),*((hipFloatComplex *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        }
    }
    else
    {
        if (v1->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_axpby_kernel<double><<< grid,block >>>
                ((double *)v1->cu_val, (double *)v2->cu_val,*((double *)a),*((double *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        } 
        else 
        {
            cu_axpby_kernel<float><<< grid,block >>>
                ((float *)v1->cu_val, (float *)v2->cu_val,*((float *)a),*((float *)b),
                 v1->traits.nrowsorig,curowfield,v1->traits.ncolsorig,cucolfield,v1->stride);
        }
    }
    
    goto out;
err:
out:
    GHOST_CALL_RETURN(ghost_cu_free(cucolfield));
    GHOST_CALL_RETURN(ghost_cu_free(curowfield));
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_scale(ghost_densemat_t *vec, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;
    
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];

    char *cucolfield = NULL, *curowfield = NULL;
    int grid = (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK);
    dim3 block (THREADSPERBLOCK/vec->traits.ncolsorig,vec->traits.ncolsorig);

    if (ghost_bitmap_weight(vec->ldmask) != vec->traits.ncolsorig || 
            ghost_bitmap_weight(vec->trmask) != vec->traits.nrowsorig) { 
        WARNING_LOG("Potentially slow SCAL operation because some rows or columns are masked out!");
        
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig),err,ret);

        ghost_densemat_mask2charfield(vec->ldmask,vec->traits.ncolsorig,colfield);
        ghost_densemat_mask2charfield(vec->trmask,vec->traits.nrowsorig,rowfield);

        GHOST_CALL_GOTO(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig),err,ret);
    }


    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_scale_kernel<hipDoubleComplex><<< grid,block >>>(
                    (hipDoubleComplex *)vec->cu_val, *(hipDoubleComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            cu_scale_kernel<hipFloatComplex><<< grid,block >>>(
                    (hipFloatComplex *)vec->cu_val, *(hipFloatComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_scale_kernel<double><<< grid,block >>>(
                    (double *)vec->cu_val, *(double *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            cu_scale_kernel<float><<< grid,block >>>(
                    (float *)vec->cu_val, *(float *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }
    goto out;

err:

out:
    GHOST_CALL_RETURN(ghost_cu_free(cucolfield));
    GHOST_CALL_RETURN(ghost_cu_free(curowfield));
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    
    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_vscale(ghost_densemat_t *vec, void *a)
{
    GHOST_FUNC_ENTER(GHOST_FUNCTYPE_MATH);
    ghost_error_t ret = GHOST_SUCCESS;

    void *d_a;
    size_t sizeofdt;
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];
    ghost_idx_t c,v=0;

    char *cucolfield = NULL, *curowfield = NULL;
    int grid = (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK);
    dim3 block (THREADSPERBLOCK/vec->traits.ncolsorig,vec->traits.ncolsorig); 
    
    ghost_datatype_size(&sizeofdt,vec->traits.datatype);
    GHOST_CALL_GOTO(ghost_cu_malloc(&d_a,vec->traits.ncolsorig*sizeofdt),err,ret);
    GHOST_CALL_GOTO(ghost_cu_memset(d_a,0,vec->traits.ncolsorig*sizeofdt),err,ret);
    
    for (c=0; c<vec->traits.ncolsorig; c++) {
        if (ghost_bitmap_isset(vec->ldmask,c)) {
            GHOST_CALL_GOTO(ghost_cu_upload(&((char *)d_a)[c*sizeofdt],&((char *)a)[v*sizeofdt],sizeofdt),err,ret);
            v++;
        }
    }
    
    if (ghost_bitmap_weight(vec->ldmask) != vec->traits.ncolsorig || 
            ghost_bitmap_weight(vec->trmask) != vec->traits.nrowsorig) { 
        
        WARNING_LOG("Potentially slow VSCALE operation because some rows or columns are masked out!");
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig),err,ret);

        ghost_densemat_mask2charfield(vec->ldmask,vec->traits.ncolsorig,colfield);
        ghost_densemat_mask2charfield(vec->trmask,vec->traits.nrowsorig,rowfield);

        GHOST_CALL_GOTO(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig),err,ret);
    }


    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<hipDoubleComplex><<< grid,block >>>(
                    (hipDoubleComplex *)vec->cu_val, (hipDoubleComplex *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            cu_vscale_kernel<hipFloatComplex><<< grid,block >>>(
                    (hipFloatComplex *)vec->cu_val, (hipFloatComplex *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_vscale_kernel<double><<< grid,block >>>(
                    (double *)vec->cu_val, (double *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            cu_vscale_kernel<float><<< grid,block >>>(
                    (float *)vec->cu_val, (float *)d_a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }

    goto out;
err:
out:
    GHOST_CALL_RETURN(ghost_cu_free(cucolfield));
    GHOST_CALL_RETURN(ghost_cu_free(curowfield));
    GHOST_CALL_RETURN(ghost_cu_free(d_a));
    GHOST_FUNC_EXIT(GHOST_FUNCTYPE_MATH);

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_fromScalar(ghost_densemat_t *vec, void *a)
{
    ghost_error_t ret = GHOST_SUCCESS;
    
    char colfield[vec->traits.ncolsorig];
    char rowfield[vec->traits.nrowsorig];

    char *cucolfield = NULL, *curowfield = NULL;
    int grid = (int)ceil((double)vec->traits.nrows/THREADSPERBLOCK);
    dim3 block (THREADSPERBLOCK/vec->traits.ncolsorig,vec->traits.ncolsorig); 

    if (ghost_bitmap_weight(vec->ldmask) != vec->traits.ncolsorig || 
            ghost_bitmap_weight(vec->trmask) != vec->traits.nrowsorig) { 
        
        WARNING_LOG("Potentially slow fromScalar operation because some rows or columns are masked out!");
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&cucolfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_malloc((void **)&curowfield,vec->traits.nrowsorig),err,ret);

        ghost_densemat_mask2charfield(vec->ldmask,vec->traits.ncolsorig,colfield);
        ghost_densemat_mask2charfield(vec->trmask,vec->traits.nrowsorig,rowfield);

        GHOST_CALL_GOTO(ghost_cu_upload(cucolfield,colfield,vec->traits.ncolsorig),err,ret);
        GHOST_CALL_GOTO(ghost_cu_upload(curowfield,rowfield,vec->traits.nrowsorig),err,ret);
    }
    
    ghost_densemat_rm_malloc(vec);
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<hipDoubleComplex><<< grid,block >>>(
                    (hipDoubleComplex *)vec->cu_val, *(hipDoubleComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            cu_fromscalar_kernel<hipFloatComplex><<< grid,block >>>(
                    (hipFloatComplex *)vec->cu_val, *(hipFloatComplex *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            cu_fromscalar_kernel<double><<< grid,block >>>(
                    (double *)vec->cu_val, *(double *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        } 
        else 
        {
            cu_fromscalar_kernel<float><<< grid,block >>>(
                    (float *)vec->cu_val, *(float *)a,
                    vec->traits.nrowsorig,curowfield,vec->traits.ncolsorig,cucolfield,vec->stride);
        }
    }
    
    goto out;
err:
out:
    GHOST_CALL_RETURN(ghost_cu_free(cucolfield));
    GHOST_CALL_RETURN(ghost_cu_free(curowfield));

    return ret;
}

extern "C" ghost_error_t ghost_densemat_rm_cu_fromRand(ghost_densemat_t *vec)
{
    ghost_error_t ret = GHOST_SUCCESS;

    ghost_densemat_t *onevec, *onevecview;
    long pid = getpid();
    double time;
    double one[] = {1.,1.};
    float fone[] = {1.,0.};
    double minusahalf[] = {-0.5,0.};
    float fminusahalf[] = {-0.5,0.};
    
    ghost_timing_wcmilli(&time);
    ghost_densemat_rm_malloc(vec);
    hiprandGenerator_t gen;
    CURAND_CALL_GOTO(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT),err,ret);
    CURAND_CALL_GOTO(hiprandSetPseudoRandomGeneratorSeed(gen,ghost_hash(int(time),clock(),pid)),err,ret);

    vec->clone(vec,&onevec,vec->traits.nrowsorig,0,vec->traits.ncolsorig,0);
    onevec->fromScalar(onevec,one);
    onevec->viewVec(onevec,&onevecview,vec->traits.nrows,ghost_bitmap_first(vec->trmask),vec->traits.ncols,ghost_bitmap_first(vec->ldmask));

    one[1] = 0.;
    void *valptr;
    ghost_densemat_t *compactvec;

    if ((vec->traits.ncolsorig != vec->traits.ncols) || (vec->traits.flags & GHOST_DENSEMAT_SCATTERED)) {
        INFO_LOG("Cloning (and compressing) vec before operation");
        vec->clone(vec,&compactvec,vec->traits.nrows,0,vec->traits.ncols,0);
    } else {
        compactvec = vec;
    }
    ghost_densemat_cu_valptr(compactvec,&valptr);

    
    if (vec->traits.datatype & GHOST_DT_COMPLEX)
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CURAND_CALL_GOTO(hiprandGenerateUniformDouble(gen,
                        (double *)(valptr),
                        compactvec->traits.ncolsorig*compactvec->traits.nrows*2),err,ret);
        } 
        else 
        {
            CURAND_CALL_GOTO(hiprandGenerateUniform(gen,
                        (float *)(valptr),
                        compactvec->traits.ncolsorig*compactvec->traits.nrows*2),err,ret);
        }
    }
    else
    {
        if (vec->traits.datatype & GHOST_DT_DOUBLE)
        {
            CURAND_CALL_GOTO(hiprandGenerateUniformDouble(gen,
                        (double *)(valptr),
                        compactvec->traits.ncolsorig*compactvec->traits.nrows),err,ret);
        } 
        else 
        {
            CURAND_CALL_GOTO(hiprandGenerateUniform(gen,
                        (float *)(valptr),
                        compactvec->traits.ncolsorig*compactvec->traits.nrows),err,ret);
        }
    }
    if (compactvec->traits.datatype & GHOST_DT_DOUBLE) {
        compactvec->axpby(compactvec,onevecview,minusahalf,one);
    } else {
        compactvec->axpby(compactvec,onevecview,fminusahalf,fone);
    }
    if (compactvec != vec) {
        vec->fromVec(vec,compactvec,0,0);
        compactvec->destroy(compactvec);
    }
    goto out;
err:
out:
    CURAND_CALL_RETURN(hiprandDestroyGenerator(gen));
    onevec->destroy(onevec);

    return ret;
}
#endif
