#include "hip/hip_runtime.h"
#include "ghost/config.h"
#undef GHOST_HAVE_MPI
#include "ghost/types.h"
#include "ghost/util.h"
#include "ghost/constants.h"
#include "ghost/vec.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>
#include <sys/types.h>
#include <unistd.h>


#include "ghost/cu_complex_helper.h"


#define THREADSPERBLOCK 256

extern hipblasHandle_t ghost_cublas_handle;

template<typename T>  
__global__ static void cu_vaxpy_kernel(T *v1, T *v2, T *a, ghost_vidx_t nrows, ghost_vidx_t nvecs, ghost_vidx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_vidx_t v;
        for (v=0; v<nvecs; v++) {
            v1[v*nrowspadded+idx] = axpy<T,T>(v1[v*nrowspadded+idx],v2[v*nrowspadded+idx],a[v]);
        }
    }
}

template<typename T>  
__global__ static void cu_vaxpby_kernel(T *v1, T *v2, T *a, T *b, ghost_vidx_t nrows, ghost_vidx_t nvecs, ghost_vidx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_vidx_t v;
        for (v=0; v<nvecs; v++) {
            v1[v*nrowspadded+idx] = axpby<T>(v2[v*nrowspadded+idx],v1[v*nrowspadded+idx],a[v],b[v]);
        }
    }
}

template<typename T>  
__global__ static void cu_axpby_kernel(T *v1, T *v2, T a, T b, ghost_vidx_t nrows, ghost_vidx_t nvecs, ghost_vidx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_vidx_t v;
        for (v=0; v<nvecs; v++) {
            v1[v*nrowspadded+idx] = axpby<T>(v2[v*nrowspadded+idx],v1[v*nrowspadded+idx],a,b);
        }
    }
}

template<typename T>  
__global__ static void cu_vscale_kernel(T *vec, T *a, ghost_vidx_t nrows, ghost_vidx_t nvecs, ghost_vidx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_vidx_t v;
        for (v=0; v<nvecs; v++) {
            vec[v*nrowspadded+idx] = scale<T>(a[v],vec[v*nrowspadded+idx]);
        }
    }
}

template<typename T>  
__global__ static void cu_fromscalar_kernel(T *vec, T a, ghost_vidx_t nrows, ghost_vidx_t nvecs, ghost_vidx_t nrowspadded)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    for (;idx < nrows; idx+=gridDim.x*blockDim.x)
    {
        ghost_vidx_t v;
        for (v=0; v<nvecs; v++) {
            vec[v*nrowspadded+idx] = a;
        }
    }
}

extern "C" void ghost_vec_cu_vaxpy(ghost_vec_t *v1, ghost_vec_t *v2, void *a)
{
    void *d_a = CU_allocDeviceMemory(v1->traits->nvecs*ghost_sizeofDataType(v1->traits->datatype));
    CU_copyHostToDevice(d_a,a,v1->traits->nvecs*ghost_sizeofDataType(v1->traits->datatype));
    if (v1->traits->datatype != v2->traits->datatype)
    {
        WARNING_LOG("Cannot VAXPY vectors with different data types");
        return;
    }

    if (v1->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
    {
        if (v1->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_vaxpy_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((hipDoubleComplex *)v1->CU_val, (hipDoubleComplex *)v2->CU_val,(hipDoubleComplex *)d_a,v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        } 
        else 
        {
            cu_vaxpy_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((hipFloatComplex *)v1->CU_val, (hipFloatComplex *)v2->CU_val,(hipFloatComplex *)d_a,v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        }
    }
    else
    {
        if (v1->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_vaxpy_kernel<double><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((double *)v1->CU_val, (double *)v2->CU_val,(double *)d_a,v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        } 
        else 
        {
            cu_vaxpy_kernel<float><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>((float *)v1->CU_val, (float *)v2->CU_val,(float *)d_a,v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        }
    }
}
    
extern "C" void ghost_vec_cu_vaxpby(ghost_vec_t *v1, ghost_vec_t *v2, void *a, void *b)
{
    void *d_a = CU_allocDeviceMemory(v1->traits->nvecs*ghost_sizeofDataType(v1->traits->datatype));
    void *d_b = CU_allocDeviceMemory(v1->traits->nvecs*ghost_sizeofDataType(v1->traits->datatype));
    CU_copyHostToDevice(d_a,a,v1->traits->nvecs*ghost_sizeofDataType(v1->traits->datatype));
    CU_copyHostToDevice(d_b,b,v1->traits->nvecs*ghost_sizeofDataType(v1->traits->datatype));
    if (v1->traits->datatype != v2->traits->datatype)
    {
        WARNING_LOG("Cannot VAXPBY vectors with different data types");
        return;
    }
    if (v1->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
    {
        if (v1->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_vaxpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipDoubleComplex *)v1->CU_val, (hipDoubleComplex *)v2->CU_val,(hipDoubleComplex *)d_a,(hipDoubleComplex *)d_b,
                 v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        } 
        else 
        {
            cu_vaxpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (hipFloatComplex *)v1->CU_val, (hipFloatComplex *)v2->CU_val,(hipFloatComplex *)d_a,(hipFloatComplex *)d_b,
                 v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        }
    }
    else
    {
        if (v1->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_vaxpby_kernel<double><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)v1->CU_val, (double *)v2->CU_val,(double *)d_a,(double *)d_b,
                    v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        } 
        else 
        {
            cu_vaxpby_kernel<float><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                (float *)v1->CU_val, (float *)v2->CU_val,(float *)d_a,(float *)d_b,
                 v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        }
    }
}

extern "C" void ghost_vec_cu_dotprod(ghost_vec_t *vec, ghost_vec_t *vec2, void *res)
{
    if (vec->traits->datatype != vec2->traits->datatype)
    {
        WARNING_LOG("Cannot DOT vectors with different data types");
        return;
    }
    
    ghost_vidx_t v;
    for (v=0; v<vec->traits->nvecs; v++)
    {
        char *v1 = &vec->CU_val[v*vec->traits->nrowspadded*ghost_sizeofDataType(vec->traits->datatype)];
        char *v2 = &vec2->CU_val[v*vec->traits->nrowspadded*ghost_sizeofDataType(vec->traits->datatype)];
        if (vec->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
        {
            if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
            {
                CUBLAS_safecall(hipblasZdotc(ghost_cublas_handle,vec->traits->nrows,
                            (const hipDoubleComplex *)v1,1,(const hipDoubleComplex *)v2,1,&((hipDoubleComplex *)res)[v]));
            } 
            else 
            {
                CUBLAS_safecall(hipblasCdotc(ghost_cublas_handle,vec->traits->nrows,
                            (const hipFloatComplex *)v1,1,(const hipFloatComplex *)v2,1,&((hipFloatComplex *)res)[v]));
            }
        }
        else
        {
            if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
            {
                CUBLAS_safecall(hipblasDdot(ghost_cublas_handle,vec->traits->nrows,
                            (const double *)v1,1,(const double *)v2,1,&((double *)res)[v]));
            } 
            else 
            {
                CUBLAS_safecall(hipblasSdot(ghost_cublas_handle,vec->traits->nrows,
                            (const float *)v1,1,(const float *)v2,1,&((float *)res)[v]));
            }
        }
    }
}

extern "C" void ghost_vec_cu_axpy(ghost_vec_t *vec, ghost_vec_t *vec2, void *a)
{
    if (vec->traits->datatype != vec2->traits->datatype)
    {
        WARNING_LOG("Cannot AXPY vectors with different data types");
        return;
    }
    if (vec->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            CUBLAS_safecall(hipblasZaxpy(ghost_cublas_handle,vec->traits->nrows,
                        (const hipDoubleComplex *)a,
                        (const hipDoubleComplex *)vec2->CU_val,1,
                        (hipDoubleComplex *)vec->CU_val,1));
        } 
        else 
        {
            CUBLAS_safecall(hipblasCaxpy(ghost_cublas_handle,vec->traits->nrows,
                        (const hipFloatComplex *)a,
                        (const hipFloatComplex *)vec2->CU_val,1,
                        (hipFloatComplex *)vec->CU_val,1));
        }
    }
    else
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            CUBLAS_safecall(hipblasDaxpy(ghost_cublas_handle,vec->traits->nrows,
                        (const double *)a,
                        (const double *)vec2->CU_val,1,
                        (double *)vec->CU_val,1));
        } 
        else 
        {
            CUBLAS_safecall(hipblasSaxpy(ghost_cublas_handle,vec->traits->nrows,
                        (const float *)a,
                        (const float *)vec2->CU_val,1,
                        (float *)vec->CU_val,1));
        }
    }
}

extern "C" void ghost_vec_cu_axpby(ghost_vec_t *v1, ghost_vec_t *v2, void *a, void *b)
{
    if (v1->traits->datatype != v2->traits->datatype)
    {
        WARNING_LOG("Cannot AXPY vectors with different data types");
        return;
    }
    if (v1->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
    {
        if (v1->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_axpby_kernel<hipDoubleComplex><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipDoubleComplex *)v1->CU_val, (hipDoubleComplex *)v2->CU_val,*((hipDoubleComplex *)a),*((hipDoubleComplex *)b),
                 v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        } 
        else 
        {
            cu_axpby_kernel<hipFloatComplex><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((hipFloatComplex *)v1->CU_val, (hipFloatComplex *)v2->CU_val,*((hipFloatComplex *)a),*((hipFloatComplex *)b),
                 v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        }
    }
    else
    {
        if (v1->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_axpby_kernel<double><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((double *)v1->CU_val, (double *)v2->CU_val,*((double *)a),*((double *)b),
                 v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        } 
        else 
        {
            cu_axpby_kernel<float><<< (int)ceil((double)v1->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>
                ((float *)v1->CU_val, (float *)v2->CU_val,*((float *)a),*((float *)b),
                 v1->traits->nrows,v1->traits->nvecs,v1->traits->nrowspadded);
        }
    }
}

extern "C" void ghost_vec_cu_scale(ghost_vec_t *vec, void *a)
{
    if (vec->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            CUBLAS_safecall(hipblasZscal(ghost_cublas_handle,vec->traits->nrows,
                        (const hipDoubleComplex *)a,
                        (hipDoubleComplex *)vec->CU_val,1));
        } 
        else 
        {
            CUBLAS_safecall(hipblasCscal(ghost_cublas_handle,vec->traits->nrows,
                        (const hipFloatComplex *)a,
                        (hipFloatComplex *)vec->CU_val,1));
        }
    }
    else
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            CUBLAS_safecall(hipblasDscal(ghost_cublas_handle,vec->traits->nrows,
                        (const double *)a,
                        (double *)vec->CU_val,1));
        } 
        else 
        {
            CUBLAS_safecall(hipblasSscal(ghost_cublas_handle,vec->traits->nrows,
                        (const float *)a,
                        (float *)vec->CU_val,1));
        }
    }
}

extern "C" void ghost_vec_cu_vscale(ghost_vec_t *vec, void *a)
{
    void *d_a = CU_allocDeviceMemory(vec->traits->nvecs*ghost_sizeofDataType(vec->traits->datatype));
    CU_copyHostToDevice(d_a,a,vec->traits->nvecs*ghost_sizeofDataType(vec->traits->datatype));
    if (vec->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_vscale_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vec->CU_val, (hipDoubleComplex *)d_a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        } 
        else 
        {
            cu_vscale_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vec->CU_val, (hipFloatComplex *)d_a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        }
    }
    else
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_vscale_kernel<double><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vec->CU_val, (double *)d_a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        } 
        else 
        {
            cu_vscale_kernel<float><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vec->CU_val, (float *)d_a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        }
    }
}

extern "C" void ghost_vec_cu_fromScalar(ghost_vec_t *vec, void *a)
{
    ghost_vec_malloc(vec);
    if (vec->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_fromscalar_kernel<hipDoubleComplex><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipDoubleComplex *)vec->CU_val, *(hipDoubleComplex *)a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        } 
        else 
        {
            cu_fromscalar_kernel<hipFloatComplex><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (hipFloatComplex *)vec->CU_val, *(hipFloatComplex *)a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        }
    }
    else
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
        {
            cu_fromscalar_kernel<double><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (double *)vec->CU_val, *(double *)a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        } 
        else 
        {
            cu_fromscalar_kernel<float><<< (int)ceil((double)vec->traits->nrows/THREADSPERBLOCK),THREADSPERBLOCK >>>(
                    (float *)vec->CU_val, *(float *)a,
                    vec->traits->nrows,vec->traits->nvecs,vec->traits->nrowspadded);
        }
    }
}

void ghost_vec_cu_fromRand(ghost_vec_t *vec)
{
    long pid = getpid();
    ghost_vec_malloc(vec);
    hiprandGenerator_t gen;
    CURAND_safecall(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_safecall(hiprandSetPseudoRandomGeneratorSeed(gen,ghost_hash(int(ghost_wctimemilli()),clock(),ghost_ompGetThreadNum())));

    ghost_vidx_t v;
    for (v=0; v<vec->traits->nvecs; v++)
    {
        if (vec->traits->datatype & GHOST_BINCRS_DT_COMPLEX)
        {
            if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
            {
                CURAND_safecall(hiprandGenerateUniformDouble(gen,
                            &((double *)(vec->CU_val))[v*vec->traits->nrowspadded],
                            vec->traits->nrows*2));
            } 
            else 
            {
                CURAND_safecall(hiprandGenerateUniform(gen,
                            &((float *)(vec->CU_val))[v*vec->traits->nrowspadded],
                            vec->traits->nrows*2));
            }
        }
        else
        {
            if (vec->traits->datatype & GHOST_BINCRS_DT_DOUBLE)
            {
                CURAND_safecall(hiprandGenerateUniformDouble(gen,
                            &((double *)(vec->CU_val))[v*vec->traits->nrowspadded],
                            vec->traits->nrows));
            } 
            else 
            {
                CURAND_safecall(hiprandGenerateUniform(gen,
                            &((float *)(vec->CU_val))[v*vec->traits->nrowspadded],
                            vec->traits->nrows));
            }
        }
    }
}
