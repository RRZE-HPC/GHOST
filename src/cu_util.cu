#include "ghost/types.h"
#include "ghost/error.h"
#include "ghost/util.h"
#include "ghost/cu_util.h"
#include "ghost/cu_sell_kernel.h"
#ifdef GHOST_HAVE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif

ghost_error ghost_cu_reduce(void *out, void *data, ghost_datatype dt, ghost_lidx n)
{
#ifdef GHOST_HAVE_CUDA
    struct hipDeviceProp_t devProp;
    int cu_device;
    GHOST_CALL_RETURN(ghost_cu_device(&cu_device));
    CUDA_CALL_RETURN(hipGetDeviceProperties(&devProp,cu_device));
    if (devProp.major < 6) {
        // call version which requires only a single block
        if (dt & GHOST_DT_COMPLEX) {
            if (dt & GHOST_DT_DOUBLE) {
                ghost_deviceReduceSum<hipDoubleComplex><<<1,1024>>>((hipDoubleComplex *)data,(hipDoubleComplex *)out,n);
            } else {
                ghost_deviceReduceSum<hipFloatComplex><<<1,1024>>>((hipFloatComplex *)data,(hipFloatComplex *)out,n);
            }
        } else {
            if (dt & GHOST_DT_DOUBLE) {
                ghost_deviceReduceSum<double><<<1,1024>>>((double *)data,(double *)out,n);
            } else {
                ghost_deviceReduceSum<float><<<1,1024>>>((float *)data,(float *)out,n);
            }
        }

    } else {
        // call version with atomic adds
        const int block = 256;
        const int grid = CEILDIV(n,block);
        if (dt & GHOST_DT_COMPLEX) {
            if (dt & GHOST_DT_DOUBLE) {
                ghost_deviceReduceSum<hipDoubleComplex><<<grid,block>>>((hipDoubleComplex *)data,(hipDoubleComplex *)out,n);
            } else {
                ghost_deviceReduceSum<hipFloatComplex><<<grid,block>>>((hipFloatComplex *)data,(hipFloatComplex *)out,n);
            }
        } else {
            if (dt & GHOST_DT_DOUBLE) {
                ghost_deviceReduceSum<double><<<grid,block>>>((double *)data,(double *)out,n);
            } else {
                ghost_deviceReduceSum<float><<<grid,block>>>((float *)data,(float *)out,n);
            }
        }
    }
#else
    UNUSED(out);
    UNUSED(data);
    UNUSED(dt);
    UNUSED(n);
#endif

    return GHOST_SUCCESS;
}
