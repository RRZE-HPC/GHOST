#include "ghost/cu_sell_kernel.h"
#include "ghost/cu_util.h"
#include "ghost/error.h"
#include "ghost/types.h"
#include "ghost/util.h"

#ifdef GHOST_HAVE_CUDA
#include <hip/hip_runtime.h>
#endif

ghost_error ghost_cu_reduce(void *out, void *data, ghost_datatype dt, ghost_lidx n)
{
#ifdef GHOST_HAVE_CUDA

    int blockSize;
    int blockCount;

    ghost_cu_deviceprop devProp;
    GHOST_CALL_RETURN(ghost_cu_deviceprop_get(&devProp));
    if (devProp.major < 6) {
        blockSize = 1024;
        blockCount = 1;
    } else {
        blockSize = 256;
        blockCount = CEILDIV(n, blockSize);
    }

    if (dt & GHOST_DT_COMPLEX) {
        if (dt & GHOST_DT_DOUBLE) {
            ghost_deviceReduceSum<hipDoubleComplex><<<blockCount, blockSize>>>((hipDoubleComplex *)data, (hipDoubleComplex *)out, n);
        } else {
            ghost_deviceReduceSum<hipFloatComplex><<<blockCount, blockSize>>>((hipFloatComplex *)data, (hipFloatComplex *)out, n);
        }
    } else {
        if (dt & GHOST_DT_DOUBLE) {
            ghost_deviceReduceSum<double><<<blockCount, blockSize>>>((double *)data, (double *)out, n);
        } else {
            ghost_deviceReduceSum<float><<<blockCount, blockSize>>>((float *)data, (float *)out, n);
        }
    }

#else
    UNUSED(out);
    UNUSED(data);
    UNUSED(dt);
    UNUSED(n);
#endif
    return GHOST_SUCCESS;
}

ghost_error ghost_cu_reduce_multiple(void *out, void *data, ghost_datatype dt, ghost_lidx n, ghost_lidx ncols)
{
#ifdef GHOST_HAVE_CUDA

    int blockSize;
    int blockCount;

    ghost_cu_deviceprop devProp;
    GHOST_CALL_RETURN(ghost_cu_deviceprop_get(&devProp));

    blockSize = 1024;
    blockCount = ncols;

    if (dt & GHOST_DT_COMPLEX) {
        if (dt & GHOST_DT_DOUBLE) {
            ghost_deviceReduceSumMultiple<hipDoubleComplex><<<blockCount, blockSize>>>((hipDoubleComplex *)data, (hipDoubleComplex *)out, n, ncols);
        } else {
            ghost_deviceReduceSumMultiple<hipFloatComplex><<<blockCount, blockSize>>>((hipFloatComplex *)data, (hipFloatComplex *)out, n, ncols);
        }
    } else {
        if (dt & GHOST_DT_DOUBLE) {
            ghost_deviceReduceSumMultiple<double><<<blockCount, blockSize>>>((double *)data, (double *)out, n, ncols);
        } else {
            ghost_deviceReduceSumMultiple<float><<<blockCount, blockSize>>>((float *)data, (float *)out, n, ncols);
        }
    }

#else
    UNUSED(out);
    UNUSED(data);
    UNUSED(dt);
    UNUSED(n);
#endif
    return GHOST_SUCCESS;
}
