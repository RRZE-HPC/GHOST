#include "hip/hip_runtime.h"
#define CUDAKERNEL
#include <ghost.h>
#include <hip/hip_runtime.h>
#include <ghost_util.h>
#include <ghost_types.h>
#include <bjds.h>
#include <ghost_cu_types_generic.h>
#include "ghost_complex.h"
#include <hip/hip_complex.h>

template<typename T>
__device__ inline void zero(T &val)
{
	val = 0.;
}

template<>
__device__ inline void zero<hipFloatComplex>(hipFloatComplex &val)
{
	val = make_hipFloatComplex(0.,0.);
}

template<>
__device__ inline void zero<hipDoubleComplex>(hipDoubleComplex &val)
{
	val = make_hipDoubleComplex(0.,0.);
}

// val += val2*val3
template<typename T, typename T2>
__device__ inline T axpy(T val, T val2, T2 val3)
{
	return val+val2*val3;
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipFloatComplex>(hipFloatComplex val, hipFloatComplex val2, hipFloatComplex val3)
{
	return hipCaddf(val,hipCmulf(val2,val3));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,double>(hipFloatComplex val, hipFloatComplex val2, double val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,float>(hipFloatComplex val, hipFloatComplex val2, float val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex(val3,0.f)));
}

template<>
__device__ inline hipFloatComplex axpy<hipFloatComplex,hipDoubleComplex>(hipFloatComplex val, hipFloatComplex val2, hipDoubleComplex val3)
{
	return hipCaddf(val,hipCmulf(val2,make_hipFloatComplex((float)(hipCreal(val3)),(float)(hipCimag(val3)))));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,double>(hipDoubleComplex val, hipDoubleComplex val2, double val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex(val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,float>(hipDoubleComplex val, hipDoubleComplex val2, float val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)val3,0.)));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipDoubleComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipDoubleComplex val3)
{
	return hipCadd(val,hipCmul(val2,val3));
}

template<>
__device__ inline hipDoubleComplex axpy<hipDoubleComplex,hipFloatComplex>(hipDoubleComplex val, hipDoubleComplex val2, hipFloatComplex val3)
{
	return hipCadd(val,hipCmul(val2,make_hipDoubleComplex((double)(hipCrealf(val3)),(double)(hipCimagf(val3)))));
}

template<>
__device__ inline double axpy<double,hipFloatComplex>(double val, double val2, hipFloatComplex val3)
{
	return val+val2*(double)hipCrealf(val3);
}


template<>
__device__ inline double axpy<double,hipDoubleComplex>(double val, double val2, hipDoubleComplex val3)
{
	return val+val2*hipCreal(val3);
}

template<>
__device__ inline float axpy<float,hipFloatComplex>(float val, float val2, hipFloatComplex val3)
{
	return val+val2*hipCrealf(val3);
}


template<>
__device__ inline float axpy<float,hipDoubleComplex>(float val, float val2, hipDoubleComplex val3)
{
	return val+val2*(float)hipCreal(val3);
}


template<typename m_t, typename v_t>  
__global__ void BJDS_kernel_CU_tmpl(v_t *lhs, v_t *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int cs = chunkstart[blockIdx.x];
		int j;
		v_t tmp;
		zero<v_t>(tmp);

		for (j=0; j<rowlen[i]; j++) {
			tmp = axpy<v_t,m_t>(tmp, rhs[col[cs + threadIdx.x + j*BJDS_LEN]], val[cs + threadIdx.x + j*BJDS_LEN]);
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] = axpy<v_t,float>(lhs[i],tmp,1.f);
		else 
			lhs[i] = tmp;
	}
}

/*template<typename m_t>  
__global__ void BJDS_kernel_CU_cvec_tmpl(hipFloatComplex *lhs, hipFloatComplex *rhs, int options, int nrows, int nrowspadded, ghost_midx_t *rowlen, ghost_midx_t *col, m_t *val, ghost_mnnz_t *chunkstart, ghost_midx_t *chunklen)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if (i<nrows) {
		int cs = chunkstart[blockIdx.x];
		int j;
		hipFloatComplex tmp = make_hipFloatComplex(0.,0.);


		for (j=0; j<rowlen[i]; j++) {
			tmp += make_hipFloatComplex(val[cs + threadIdx.x + j*BJDS_LEN])  // TODO cast besser machen
				* rhs[col[cs + threadIdx.x + j*BJDS_LEN]];
		}
		if (options & GHOST_SPMVM_AXPY)
			lhs[i] += tmp;
		else 
			lhs[i] = tmp;
	}
}*/

extern "C" void dd_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< double,double > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(double *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void ds_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< double,float > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(double *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void dc_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options){ return BJDS_kernel_CU_tmpl< double > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipFloatComplex*)lhs->CU_val,(hipFloatComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(double *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void dz_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< double,hipDoubleComplex > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(double *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void sd_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< float,double > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(float *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void ss_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< float,float > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(float *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void sc_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< float,hipComplex > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(float *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void sz_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< float,hipDoubleComplex > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(float *)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void cd_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipFloatComplex,double > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void cs_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipFloatComplex,float > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void cc_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipFloatComplex,hipComplex > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void cz_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipFloatComplex,hipDoubleComplex > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void zd_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipDoubleComplex,double > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((double *)lhs->CU_val,(double *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipDoubleComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void zs_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipDoubleComplex,float > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((float *)lhs->CU_val,(float *)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipDoubleComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void zc_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipDoubleComplex,hipFloatComplex > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipComplex*)lhs->CU_val,(hipComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipDoubleComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

extern "C" void zz_BJDS_kernel_CU(ghost_mat_t *mat, ghost_vec_t *lhs, ghost_vec_t *rhs, int options)
{ return BJDS_kernel_CU_tmpl< hipDoubleComplex,hipDoubleComplex > <<<ceil(BJDS(mat)->cumat->nrows/256.),256>>> ((hipDoubleComplex*)lhs->CU_val,(hipDoubleComplex*)rhs->CU_val,options,BJDS(mat)->cumat->nrows,BJDS(mat)->cumat->nrowsPadded,BJDS(mat)->cumat->rowLen,BJDS(mat)->cumat->col,(hipDoubleComplex*)BJDS(mat)->cumat->val,BJDS(mat)->cumat->chunkStart,BJDS(mat)->cumat->chunkLen); }

