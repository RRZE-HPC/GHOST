#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <hip/hip_runtime_api.h>
#include "cudamacros.h"


/* *********** DEVICE SELECTION ************************* */

extern "C" void getDeviceInfo( int rank, int size, const char* hostname) {
	int deviceCount, device;
	hipDeviceProp_t deviceProp;

	hipGetDeviceCount(&deviceCount);

	if ( 0 == rank ) {
		printf("## rank %i/%i on %s --\t Device Test: No. Cards: %d\n", 
				rank, size-1, hostname, deviceCount);
		for( device = 0; device < deviceCount; ++device) {
			hipGetDeviceProperties(&deviceProp, device);
			printf("## rank %i/%i on %s --\t Device %d: %s\n", 
					rank, size-1, hostname, device, deviceProp.name);
		}
	}
}


extern "C" int selectDevice( int rank, int size, const char* hostname ) {
	int deviceCount, takedevice, device;
	hipDeviceProp_t deviceProp;

	hipGetDeviceCount(&deviceCount);

	takedevice = (rank%deviceCount);
	hipSetDevice(takedevice);
	hipGetDevice(&device);
	hipGetDeviceProperties(&deviceProp, device);

	printf("rank %i/%i on %s --\t Selecting Device %d: %s\n", 
			rank, size-1, hostname, device, deviceProp.name);

	return device;
}


/* *********** KERNEL LAUNCH PARAMETERS ***************** */

typedef struct {
	int gridDim;
	int blockDim;
} KERNEL_LAUNCHER;

KERNEL_LAUNCHER _launcher_;

extern "C" void setKernelDims( const int gridDim, const int blockDim ) {

	/* set kernel launch parameters in global object _launcher_;
	 * _launcher_ used for all subsequent CUDA kernels */

	_launcher_.gridDim  = gridDim;
	_launcher_.blockDim = blockDim;
}


/* *********** TEXTURE CACHE *************************** */

#ifdef TEXCACHE
texture<int2, 1, hipReadModeElementType> texRef;
const textureReference* texRefPtr;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();

extern "C" void prepareTexCacheRhs(double * rhsVec, size_t memSize) 
{
	safecall(hipGetTextureReference( &texRefPtr, HIP_SYMBOL("texRef" )));
	safecall(hipBindTexture( 0, texRefPtr, rhsVec, &channelDesc, memSize ));
}
#endif

#ifdef COLSTARTTC
texture<int, 1, hipReadModeElementType> colStartTexRef;
const textureReference* colStartTexRefPtr;
hipChannelFormatDesc colStartChannelDesc = hipCreateChannelDesc<int>();

extern "C" void prepareTexCacheCS(int * colStartVec, size_t memSize) 
{
	safecall(hipGetTextureReference( &colStartTexRefPtr, HIP_SYMBOL("colStartTexRef" )));
	safecall(hipBindTexture( 0, colStartTexRefPtr, colStartVec, &colStartChannelDesc, memSize ));
}
#endif




/* *********** CUDA MEMORY **************************** */

extern "C" void* allocDeviceMemory( size_t bytesize ) {
	char* mem = NULL;
	safecall(hipMalloc( (void**)&mem, bytesize ));

	return (void*)mem;
}

extern "C" void* allocHostMemory( size_t bytesize ) {
	char* mem = NULL;
	safecall(hipHostAlloc( (void**)&mem, bytesize, 0 ));
	//mem = (char*) malloc( bytesize );
	//if( NULL == mem ) printf("failed to allocate %lu bytes of memory\n",bytesize);

	return (void*)mem;
}


extern "C" void copyDeviceToHost( void* hostmem, void* devmem, size_t bytesize ) {
	safecall(hipMemcpy( hostmem, devmem, bytesize, hipMemcpyDeviceToHost ));
}

extern "C" void copyHostToDevice( void* devmem, void* hostmem, size_t bytesize ) {
	safecall(hipMemcpy( devmem, hostmem, bytesize, hipMemcpyHostToDevice ));
}


extern "C" void freeDeviceMemory( void* mem ) {
	safecall(hipFree( mem ));
}

extern "C" void freeHostMemory( void* mem ) {
	safecall(hipHostFree( mem ));
}

#ifdef TEXCACHE
static __inline__ __device__ double fetch_double(texture<int2, 1> t, int i)
{
	int2 v = tex1Dfetch(t,i);
	return __hiloint2double(v.y, v.x);
}
#endif

/* *********** KERNEL **************************** */
template<bool add> __global__ void __ELRkernel__(  ELRkernelArgs args ) {

	int idx, i, idcol, max;
	double svalue, value;

	for( idx = blockIdx.x * blockDim.x + threadIdx.x; idx < args.N; idx += gridDim.x * blockDim.x ) {
		svalue = 0.0;
		max = args.rowLen[idx];
		for( i = 0; i < max; ++i) {
			value = args.val[i*args.pad+idx];
			idcol = args.col[i*args.pad+idx];
			svalue += value * RHS(idcol);
		}
		if (add)
			args.resVec[idx] += svalue;
		else
			args.resVec[idx] = svalue;
	}
}
template<bool add> __global__ void __pJDSkernel__(  pJDSkernelArgs args ) {

	int idx, i, idcol, max;
	double svalue, value;

	for( idx = blockIdx.x * blockDim.x + threadIdx.x; idx < args.N; idx += gridDim.x * blockDim.x ) {
		svalue = 0.0;
		max = args.rowLen[idx];
		for( i = 0; i < max; ++i) {
			value = args.val[COLSTART(i)+idx];
			idcol = args.col[COLSTART(i)+idx];
			svalue += value * RHS(idcol);
		}
		if (add)
			args.resVec[idx] += svalue;
		else
			args.resVec[idx] = svalue;
	}
}

extern "C" void cudaKernel( void* args, bool add, bool elr) {

	if (elr) {
		if (add)
			__ELRkernel__<true> <<< _launcher_.gridDim, _launcher_.blockDim >>> ( *((ELRkernelArgs *)(args)) );
		else
			__ELRkernel__<false> <<< _launcher_.gridDim, _launcher_.blockDim >>> ( *((ELRkernelArgs *)(args)) );
	} else {
		if (add)
			__pJDSkernel__<true> <<< _launcher_.gridDim, _launcher_.blockDim >>> ( *((pJDSkernelArgs *)(args)) );
		else
			__pJDSkernel__<false> <<< _launcher_.gridDim, _launcher_.blockDim >>> ( *((pJDSkernelArgs *)(args)) );
	}

	safecall(hipDeviceSynchronize());
	safecall(hipGetLastError());
}
